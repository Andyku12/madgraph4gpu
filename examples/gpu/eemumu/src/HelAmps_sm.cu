#include "hip/hip_runtime.h"
#ifndef gHelAmps_sm
#define gHelAmps_sm

#include "HelAmps_sm.h"
#include <cmath>
#include <complex>
#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#define gpuErrchk2(ans)                                                        \
  { gpuAssert2((ans), __FILE__, __LINE__); }

__device__ void gpuAssert2(hipError_t code, const char *file, int line,
                           bool abort = true) {
  if (code != hipSuccess) {
    printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
  }
}

extern __constant__ int cHel[16][4];
extern __constant__ double cMME[4];
extern __constant__ int cPerm[4];
extern __constant__ thrust::complex<double> cIPC[3];
extern __constant__ double cIPD[2];
// extern __shared__ thrust::complex<double> sw[6][6];

namespace gMG5_sm {

#ifdef DEBUG
__device__ void debugMsg(const char *msg) {
  printf("%i.%i-%s ", blockIdx.x, threadIdx.x, msg);
}
#endif

// later
/*
void CPPProcess::sigmaKin(int ncomb, bool (&goodhel)[16], int &ntry,
                          int &sum_hel, int &ngood, int (&igood)[16],
                          int &jhel) {
                    */
__global__ void sigmaKin(hipPitchedPtr tp, double *meDevPtr, size_t mePitch,
                         bool debug, bool verbose) {

  // for (int xx = 0; xx < 384; ++xx) {

  int nprocesses = 1;

  int dim = blockIdx.x * blockDim.x + threadIdx.x;

  char *devPtr = (char *)tp.ptr;
  size_t dpt = tp.pitch;
  size_t slicePitch = dpt * 4;

  char *dps = devPtr + dim * slicePitch;
  double *matrix_element = (double *)((char *)meDevPtr + dim * mePitch);

  thrust::complex<double> amp[2];
  double t[1];

  // <later>
  // Local variables and constants
  const int ncomb = 16;
  static bool goodhel[ncomb] = {ncomb * false};
  static int ntry = 0, ngood = 0;
  static int igood[ncomb];
  // </later>

  // Denominators: spins, colors and identical particles
  const int denominators[1] = {4}; // nprocesses

  ntry = ntry + 1;

  // Reset the matrix elements
  for (int i = 0; i < nprocesses; ++i) { // nprocesses
    matrix_element[i] = 0.;
  }

  // sr fixme // better to run the first n calculations serial?
  // if (sum_hel == 0 || ntry < 10) {
  // Calculate the matrix element for all helicities

  for (int ihel = 0; ihel < ncomb; ihel++) {
    if (goodhel[ihel] || ntry < 2) {

      calculate_wavefunctions(ihel, dps, dpt, amp, debug, verbose);
      matrix_1_epem_mupmum(t[0], amp);

      double tsum = 0;
      for (int iproc = 0; iproc < nprocesses; iproc++) {
        matrix_element[iproc] += t[iproc];
        tsum += t[iproc];
      }

      // Store which helicities give non-zero result
      if (tsum != 0. && !goodhel[ihel]) {
        goodhel[ihel] = true;
        ngood++;
        igood[ngood] = ihel;
      }
    }
  }

  for (int i = 0; i < nprocesses; ++i) {
    matrix_element[i] /= denominators[i];
  }
  // }

  // printf("%d - %e\n", dim, t[0]);
}

// --> calculate multi-dimensional amp
__device__ void matrix_1_epem_mupmum(double &matrix,
                                     thrust::complex<double> amp[2]) {
  int i, j;
  // Local variables
  // const int ngraphs = 2;
  const int ncolor = 1;
  thrust::complex<double> ztemp;
  thrust::complex<double> jamp[ncolor];
  // The color matrix;
  static const double denom[ncolor] = {1};
  static const double cf[ncolor][ncolor] = {{1}};

  // Calculate color flows
  jamp[0] = -amp[0] - amp[1];

  // Sum and square the color flows to get the matrix element
  matrix = 0;
  for (i = 0; i < ncolor; i++) {
    ztemp = 0.;
    for (j = 0; j < ncolor; j++)
      ztemp = ztemp + cf[i][j] * jamp[j];
    matrix = matrix + (ztemp * conj(jamp[i])).real() / denom[i];
  }

  // Store the leading color flows for choice of color
  // sr fixme // maybe this needs to go outside the loop? does it need a
  // dimension?
  /* sr fixme
  for (i = 0; i < ncolor; i++)
    jamp2[0][i] += (jamp[i] * conj(jamp[i])).real();
  */
}

__device__ void calculate_wavefunctions(int ihel, char *dps, size_t dpt,
                                        thrust::complex<double> amp[2],
                                        bool debug, bool verbose) {
#ifdef DEBUG
  debugMsg("%>");
  if (debug) {
    printf("\n\nblock (%i / %i), thread (%i)\n\n", blockIdx.x, blockDim.x,
           threadIdx.x);
  }
#endif

  double ZERO = 0.00;
  thrust::complex<double> sw[6][6];

  // Calculate all wavefunctions
  oxxxxx((double *)(dps + cPerm[0] * dpt), cMME[0], cHel[ihel][0], -1, sw[0]);
  ixxxxx((double *)(dps + cPerm[1] * dpt), cMME[1], cHel[ihel][1], +1, sw[1]);
  ixxxxx((double *)(dps + cPerm[2] * dpt), cMME[2], cHel[ihel][2], -1, sw[2]);
  oxxxxx((double *)(dps + cPerm[3] * dpt), cMME[3], cHel[ihel][3], +1, sw[3]);
  FFV1P0_3(sw[1], sw[0], cIPC[0], ZERO, ZERO, sw[4]);
  FFV2_4_3(sw[1], sw[0], -cIPC[1], cIPC[2], cIPD[0], cIPD[1], sw[5]);
  // Calculate all amplitudes
  // Amplitude(s) for diagram number 0
  FFV1_0(sw[2], sw[3], sw[4], cIPC[0], &amp[0]);
  FFV2_4_0(sw[2], sw[3], sw[5], -cIPC[1], cIPC[2], &amp[1]);

#ifdef DEBUG
  if (debug) {
    printf("\n\n >>> DEBUG >>> DEBUG >>> DEBUG >>>\n");

    printf("\nHelicities: %d %d %d %d\n", cHel[ihel][0], cHel[ihel][1],
           cHel[ihel][2], cHel[ihel][3]);

    printf("\nMomenta:\n");
    for (int i = 0; i < 4; ++i) {
      printf("%i %e %e %e %e\n", i, dp[cPerm[i]][0], dp[cPerm[i]][1],
             dp[cPerm[i]][2], dp[cPerm[i]][3]);
    }

    printf("\nMasses: %e, %e, %e, %e\n", cMME[0], cMME[1], cMME[2], cMME[3]);

    printf("\nAmplitudes: (%e, %e), (%e, %e)\n", amp[0].real(), amp[0].imag(),
           amp[1].real(), amp[1].imag());

    printf("\nWavefuncs:\n");
    for (int i = 0; i < 6; ++i) {
      printf("%i ", i);
      for (int j = 0; j < 6; ++j) {
        double re = sw[i][j].real(), im = sw[i][j].imag();
        if (re == 0 && im == 0) {
          printf("0, ");
        } else {
          printf("(%e, %e), ", re, im);
        }
      }
      printf("\n");
    }

    printf("\n\n <<< DEBUG <<< DEBUG <<< DEBUG <<<\n\n");
  }
  debugMsg("<%");
#endif
}

__device__ void ixxxxx(double p[4], double fmass, int nhel, int nsf,
                       thrust::complex<double> fi[6]) {
#ifdef DEBUG
  debugMsg("b>");
#endif
  thrust::complex<double> chi[2];
  double sqp0p3;
  int nh;
  fi[0] = thrust::complex<double>(-p[0] * nsf, -p[3] * nsf);
  fi[1] = thrust::complex<double>(-p[1] * nsf, -p[2] * nsf);
  nh = nhel * nsf;
  if (p[1] == 0.0 and p[2] == 0.0 and p[3] < 0.0) {
    sqp0p3 = 0.0;
  } else {
    sqp0p3 = sqrt(max(p[0] + p[3], 0.0)) * nsf;
  }
  chi[0] = thrust::complex<double>(sqp0p3, 0.0);
  if (sqp0p3 == 0.0) {
    chi[1] = thrust::complex<double>(-nhel * sqrt(2.0 * p[0]), 0.0);
  } else {
    chi[1] = thrust::complex<double>(nh * p[1], p[2]) / sqp0p3;
  }
  if (nh == 1) {
    fi[2] = thrust::complex<double>(0.0, 0.0);
    fi[3] = thrust::complex<double>(0.0, 0.0);
    fi[4] = chi[0];
    fi[5] = chi[1];
  } else {
    fi[2] = chi[1];
    fi[3] = chi[0];
    fi[4] = thrust::complex<double>(0.0, 0.0);
    fi[5] = thrust::complex<double>(0.0, 0.0);
  }
#ifdef DEBUG
  debugMsg("<b");
#endif
  return;
}

__device__ void oxxxxx(double p[4], double fmass, int nhel, int nsf,
                       thrust::complex<double> fo[6]) {
#ifdef DEBUG
  debugMsg("a>");
#endif
  thrust::complex<double> chi[2];
  double sqp0p3;
  int nh;
  fo[0] = thrust::complex<double>(p[0] * nsf, p[3] * nsf);
  fo[1] = thrust::complex<double>(p[1] * nsf, p[2] * nsf);
  nh = nhel * nsf;
  if ((p[1] == 0.00) and (p[2] == 0.00) and (p[3] < 0.00)) {
    sqp0p3 = 0.00;
  } else {
    sqp0p3 = sqrt(max(p[0] + p[3], 0.00)) * nsf;
  }
  chi[0] = thrust::complex<double>(sqp0p3, 0.00);
  if (sqp0p3 == 0.000) {
    chi[1] = thrust::complex<double>(-nhel, 0.00) * sqrt(2.0 * p[0]);
  } else {
    chi[1] = thrust::complex<double>(nh * p[1], -p[2]) / sqp0p3;
  }
  if (nh == 1) {
    fo[2] = chi[0];
    fo[3] = chi[1];
    fo[4] = thrust::complex<double>(0.00, 0.00);
    fo[5] = thrust::complex<double>(0.00, 0.00);
  } else {
    fo[2] = thrust::complex<double>(0.00, 0.00);
    fo[3] = thrust::complex<double>(0.00, 0.00);
    fo[4] = chi[1];
    fo[5] = chi[0];
  }
#ifdef DEBUG
  debugMsg("<a");
#endif
  return;
}

__device__ void FFV1_0(thrust::complex<double> F1[],
                       thrust::complex<double> F2[],
                       thrust::complex<double> V3[],
                       thrust::complex<double> COUP,
                       thrust::complex<double> *vertex) {
#ifdef DEBUG
  debugMsg("g>");
#endif
  thrust::complex<double> cI = thrust::complex<double>(0., 1.);
  thrust::complex<double> TMP2 =
      (F1[2] * (F2[4] * (V3[2] + V3[5]) + F2[5] * (V3[3] + cI * (V3[4]))) +
       (F1[3] * (F2[4] * (V3[3] - cI * (V3[4])) + F2[5] * (V3[2] - V3[5])) +
        (F1[4] * (F2[2] * (V3[2] - V3[5]) - F2[3] * (V3[3] + cI * (V3[4]))) +
         F1[5] * (F2[2] * (+cI * (V3[4]) - V3[3]) + F2[3] * (V3[2] + V3[5])))));
  (*vertex) = COUP * -cI * TMP2;
#ifdef DEBUG
  debugMsg("<g");
#endif
}

__device__ void FFV2_3(thrust::complex<double> F1[],
                       thrust::complex<double> F2[],
                       thrust::complex<double> COUP, double M3, double W3,
                       thrust::complex<double> V3[]) {
#ifdef DEBUG
  debugMsg("e>");
#endif
  thrust::complex<double> cI = thrust::complex<double>(0., 1.);
  thrust::complex<double> denom;
  thrust::complex<double> TMP1;
  double P3[4];
  double OM3;
  OM3 = 0.;
  if (M3 != 0.)
    OM3 = 1. / (M3 * M3);
  V3[0] = +F1[0] + F2[0];
  V3[1] = +F1[1] + F2[1];
  P3[0] = -V3[0].real();
  P3[1] = -V3[1].real();
  P3[2] = -V3[1].imag();
  P3[3] = -V3[0].imag();
  TMP1 = (F1[2] * (F2[4] * (P3[0] + P3[3]) + F2[5] * (P3[1] + cI * (P3[2]))) +
          F1[3] * (F2[4] * (P3[1] - cI * (P3[2])) + F2[5] * (P3[0] - P3[3])));
  denom = COUP / ((P3[0] * P3[0]) - (P3[1] * P3[1]) - (P3[2] * P3[2]) -
                  (P3[3] * P3[3]) - M3 * (M3 - cI * W3));
  V3[2] = denom * (-cI) * (F1[2] * F2[4] + F1[3] * F2[5] - P3[0] * OM3 * TMP1);
  V3[3] = denom * (-cI) * (-F1[2] * F2[5] - F1[3] * F2[4] - P3[1] * OM3 * TMP1);
  V3[4] = denom * (-cI) *
          (-cI * (F1[2] * F2[5]) + cI * (F1[3] * F2[4]) - P3[2] * OM3 * TMP1);
  V3[5] = denom * (-cI) * (F1[3] * F2[5] - F1[2] * F2[4] - P3[3] * OM3 * TMP1);
#ifdef DEBUG
  debugMsg("<e");
#endif
}

__device__ void FFV4_3(thrust::complex<double> F1[],
                       thrust::complex<double> F2[],
                       thrust::complex<double> COUP, double M3, double W3,
                       thrust::complex<double> V3[]) {
#ifdef DEBUG
  debugMsg("f>");
#endif
  thrust::complex<double> cI = thrust::complex<double>(0., 1.);
  thrust::complex<double> denom;
  thrust::complex<double> TMP1;
  double P3[4];
  double OM3;
  thrust::complex<double> TMP4;
  OM3 = 0.;
  if (M3 != 0.)
    OM3 = 1. / (M3 * M3);
  V3[0] = +F1[0] + F2[0];
  V3[1] = +F1[1] + F2[1];
  P3[0] = -V3[0].real();
  P3[1] = -V3[1].real();
  P3[2] = -V3[1].imag();
  P3[3] = -V3[0].imag();
  TMP4 = (F1[4] * (F2[2] * (P3[0] - P3[3]) - F2[3] * (P3[1] + cI * (P3[2]))) +
          F1[5] * (F2[2] * (+cI * (P3[2]) - P3[1]) + F2[3] * (P3[0] + P3[3])));
  TMP1 = (F1[2] * (F2[4] * (P3[0] + P3[3]) + F2[5] * (P3[1] + cI * (P3[2]))) +
          F1[3] * (F2[4] * (P3[1] - cI * (P3[2])) + F2[5] * (P3[0] - P3[3])));
  denom = COUP / ((P3[0] * P3[0]) - (P3[1] * P3[1]) - (P3[2] * P3[2]) -
                  (P3[3] * P3[3]) - M3 * (M3 - cI * W3));
  V3[2] = denom * (-2. * cI) *
          (OM3 * -1. / 2. * P3[0] * (TMP1 + 2. * (TMP4)) +
           (+1. / 2. * (F1[2] * F2[4] + F1[3] * F2[5]) + F1[4] * F2[2] +
            F1[5] * F2[3]));
  V3[3] = denom * (-2. * cI) *
          (OM3 * -1. / 2. * P3[1] * (TMP1 + 2. * (TMP4)) +
           (-1. / 2. * (F1[2] * F2[5] + F1[3] * F2[4]) + F1[4] * F2[3] +
            F1[5] * F2[2]));
  V3[4] = denom * 2. * cI *
          (OM3 * 1. / 2. * P3[2] * (TMP1 + 2. * (TMP4)) +
           (+1. / 2. * cI * (F1[2] * F2[5]) - 1. / 2. * cI * (F1[3] * F2[4]) -
            cI * (F1[4] * F2[3]) + cI * (F1[5] * F2[2])));
  V3[5] = denom * 2. * cI *
          (OM3 * 1. / 2. * P3[3] * (TMP1 + 2. * (TMP4)) +
           (+1. / 2. * (F1[2] * F2[4]) - 1. / 2. * (F1[3] * F2[5]) -
            F1[4] * F2[2] + F1[5] * F2[3]));
#ifdef DEBUG
  debugMsg("<f");
#endif
}

__device__ void FFV2_4_3(thrust::complex<double> F1[],
                         thrust::complex<double> F2[],
                         thrust::complex<double> COUP1,
                         thrust::complex<double> COUP2, double M3, double W3,
                         thrust::complex<double> V3[]) {
#ifdef DEBUG
  debugMsg("d>");
#endif
  int i;
  thrust::complex<double> *Vtmp;
  gpuErrchk2(hipMalloc(&Vtmp, 6 * sizeof(thrust::complex<double>)));
  *Vtmp = thrust::complex<double>(0, 0);
  FFV2_3(F1, F2, COUP1, M3, W3, V3);
  FFV4_3(F1, F2, COUP2, M3, W3, Vtmp);
  i = 2;
  while (i < 6) {
    V3[i] = V3[i] + Vtmp[i];
    i++;
  }
  gpuErrchk2(hipFree(Vtmp));
#ifdef DEBUG
  debugMsg("<d");
#endif
}

__device__ void FFV1P0_3(thrust::complex<double> F1[],
                         thrust::complex<double> F2[],
                         thrust::complex<double> COUP, double M3, double W3,
                         thrust::complex<double> V3[]) {
#ifdef DEBUG
  debugMsg("c>");
#endif
  thrust::complex<double> cI = thrust::complex<double>(0., 1.);
  double P3[4];
  thrust::complex<double> denom;
  V3[0] = +F1[0] + F2[0];
  V3[1] = +F1[1] + F2[1];
  P3[0] = -V3[0].real();
  P3[1] = -V3[1].real();
  P3[2] = -V3[1].imag();
  P3[3] = -V3[0].imag();
  denom = COUP / ((P3[0] * P3[0]) - (P3[1] * P3[1]) - (P3[2] * P3[2]) -
                  (P3[3] * P3[3]) - M3 * (M3 - cI * W3));
  V3[2] = denom * (-cI) *
          (F1[2] * F2[4] + F1[3] * F2[5] + F1[4] * F2[2] + F1[5] * F2[3]);
  V3[3] = denom * (-cI) *
          (F1[4] * F2[3] + F1[5] * F2[2] - F1[2] * F2[5] - F1[3] * F2[4]);
  V3[4] = denom * (-cI) *
          (-cI * (F1[2] * F2[5] + F1[5] * F2[2]) +
           cI * (F1[3] * F2[4] + F1[4] * F2[3]));
  V3[5] = denom * (-cI) *
          (F1[3] * F2[5] + F1[4] * F2[2] - F1[2] * F2[4] - F1[5] * F2[3]);
#ifdef DEBUG
  debugMsg("<c");
#endif
}

__device__ void FFV4_0(thrust::complex<double> F1[],
                       thrust::complex<double> F2[],
                       thrust::complex<double> V3[],
                       thrust::complex<double> COUP,
                       thrust::complex<double> *vertex) {
#ifdef DEBUG
  debugMsg("j>");
#endif
  thrust::complex<double> cI = thrust::complex<double>(0., 1.);
  thrust::complex<double> TMP0, TMP3;
  TMP0 = (F1[2] * (F2[4] * (V3[2] + V3[5]) + F2[5] * (V3[3] + cI * (V3[4]))) +
          F1[3] * (F2[4] * (V3[3] - cI * (V3[4])) + F2[5] * (V3[2] - V3[5])));
  TMP3 = (F1[4] * (F2[2] * (V3[2] - V3[5]) - F2[3] * (V3[3] + cI * (V3[4]))) +
          F1[5] * (F2[2] * (+cI * (V3[4]) - V3[3]) + F2[3] * (V3[2] + V3[5])));
  (*vertex) = COUP * (-1.) * (+cI * (TMP0) + 2. * cI * (TMP3));
#ifdef DEBUG
  debugMsg("<j");
#endif
}

__device__ void FFV2_0(thrust::complex<double> F1[],
                       thrust::complex<double> F2[],
                       thrust::complex<double> V3[],
                       thrust::complex<double> COUP,
                       thrust::complex<double> *vertex) {
#ifdef DEBUG
  debugMsg("i>");
#endif
  thrust::complex<double> cI = thrust::complex<double>(0., 1.);
  thrust::complex<double> TMP0;
  TMP0 = (F1[2] * (F2[4] * (V3[2] + V3[5]) + F2[5] * (V3[3] + cI * (V3[4]))) +
          F1[3] * (F2[4] * (V3[3] - cI * (V3[4])) + F2[5] * (V3[2] - V3[5])));
  (*vertex) = COUP * -cI * TMP0;
#ifdef DEBUG
  debugMsg("<i");
#endif
}

__device__ void
FFV2_4_0(thrust::complex<double> F1[], thrust::complex<double> F2[],
         thrust::complex<double> V3[], thrust::complex<double> COUP1,
         thrust::complex<double> COUP2, thrust::complex<double> *vertex) {
#ifdef DEBUG
  debugMsg("h>");
#endif
  thrust::complex<double> tmp;
  FFV2_0(F1, F2, V3, COUP1, vertex);
  FFV4_0(F1, F2, V3, COUP2, &tmp);
  (*vertex) = (*vertex) + tmp;
#ifdef DEBUG
  debugMsg("<h");
#endif
}

} // namespace gMG5_sm

#endif // gHelAmps_sm
