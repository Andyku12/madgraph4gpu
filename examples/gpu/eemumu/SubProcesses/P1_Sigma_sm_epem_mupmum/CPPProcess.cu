#include "hip/hip_runtime.h"
//==========================================================================
// This file has been automatically generated for C++ Standalone by
// MadGraph5_aMC@NLO v. 2.7.0, 2020-01-20
// By the MadGraph5_aMC@NLO Development Team
// Visit launchpad.net/madgraph5 and amcatnlo.web.cern.ch
//==========================================================================

#include "CPPProcess.h"
// #include "HelAmps_sm.h"
#include "HelAmps_sm.h"

#include <iomanip> // setw
#include <iostream>
#include <thrust/complex.h>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

CPPProcess::CPPProcess(bool verbose, bool debug)
    : m_verbose(verbose), m_debug(debug), mME(4, 0.00) {

  m = new processMem();

  // mm
  // mME
  static double tmpmME[4] = {0.00, 0.00, 0.00, 0.00};
  gpuErrchk(hipMallocManaged(&m->mME, 4 * sizeof(double)));
  gpuErrchk(hipMemcpy((void *)m->mME, (void *)tmpmME, 4 * sizeof(double),
                       hipMemcpyHostToHost));

  // mm
  // p
  ptemp = (double **)malloc(m->tnioparticles * sizeof(double));
  for (int i = 0; i < 4; ++i) {
    ptemp[i] = (double *)malloc(4 * sizeof(double));
  }
  gpuErrchk(hipMallocManaged(&m->tp, m->tnioparticles * sizeof(double)));
  for (int i = 0; i < 4; ++i) {
    gpuErrchk(hipMallocManaged(&m->tp[i], 4 * sizeof(double)));
  }

  // amp
  gpuErrchk(hipMallocManaged(&m->tamp, m->tnamplitudes *
                                            sizeof(thrust::complex<double>)));

  // w
  gpuErrchk(
      hipMallocManaged(&m->tw, m->twrows * sizeof(thrust::complex<double>)));
  for (int i = 0; i < m->twrows; ++i) {
    gpuErrchk(hipMallocManaged(
        &m->tw[i], m->tnwavefuncs * sizeof(thrust::complex<double>)));
    for (int y = 0; y < m->twrows; ++y) {
      m->tw[i][y] = thrust::complex<double>(0.00, 0.00);
    }
  }

  // mm
  // helicities
  // Helicities for the process
  static int helicities[m->tncomb][m->tnexternal] = {
      {-1, -1, -1, -1}, {-1, -1, -1, 1}, {-1, -1, 1, -1}, {-1, -1, 1, 1},
      {-1, 1, -1, -1},  {-1, 1, -1, 1},  {-1, 1, 1, -1},  {-1, 1, 1, 1},
      {1, -1, -1, -1},  {1, -1, -1, 1},  {1, -1, 1, -1},  {1, -1, 1, 1},
      {1, 1, -1, -1},   {1, 1, -1, 1},   {1, 1, 1, -1},   {1, 1, 1, 1}};
  gpuErrchk(hipMallocManaged(&m->thelicities,
                              m->tncomb * m->tnexternal * sizeof(int)));
  for (int i = 0; i < m->tncomb; ++i) {
    gpuErrchk(hipMemcpy(&m->thelicities[i], &helicities[i],
                         m->tnexternal * sizeof(int), hipMemcpyHostToHost));
  }

  // mm
  // perm
  static int perm[m->tnexternal];
  for (int i = 0; i < m->tnexternal; i++) {
    perm[i] = i;
  }
  gpuErrchk(hipMallocManaged(&m->tperm, m->tnexternal * sizeof(int)));
  gpuErrchk(hipMemcpy((void *)m->tperm, (void *)perm,
                       m->tnexternal * sizeof(int), hipMemcpyHostToHost));
}

CPPProcess::~CPPProcess() {}

void CPPProcess::setMomenta(std::vector<double *> &momenta) {

  for (int i = 0; i < m->tnioparticles; ++i) {
    gpuErrchk(hipMemcpy((void *)m->tp[i], (void *)momenta[i],
                         4 * sizeof(double), hipMemcpyHostToHost));
    // memcpy(ptemp[i], momenta[i], 4 * sizeof(double));
  }

  /*
  for (std::vector<double *>::iterator it = momenta.begin();
       it != momenta.end(); ++it) {
    double *tmp;
    hipMallocManaged(&tmp, 4 * sizeof(double));
    hipMemcpy(tmp, *it, 4 * sizeof(double), hipMemcpyHostToHost);
    p.push_back(tmp);
  }
  */
}

const std::vector<double> &CPPProcess::getMasses() const { return mME; }

//==========================================================================
// Class member functions for calculating the matrix elements for
// Process: e+ e- > mu+ mu- WEIGHTED<=4 @1

//--------------------------------------------------------------------------
// Initialize process.

void CPPProcess::initProc(std::string param_card_name) {

  // Instantiate the model class and set parameters that stay fixed during run
  pars = Parameters_sm::getInstance();
  SLHAReader slha(param_card_name, m_verbose);
  pars->setIndependentParameters(slha);
  pars->setIndependentCouplings();
  if (m_verbose) {
    pars->printIndependentParameters();
    pars->printIndependentCouplings();
  }
  jamp2[0] = new double[1];
}

void CPPProcess::resetGPUMemory() {

  for (std::vector<double *>::iterator it = p.begin(); it != p.end(); ++it) {
    gpuErrchk(hipFree(*it));
  }
  p.clear();
}

//--------------------------------------------------------------------------
// Evaluate |M|^2, part independent of incoming flavour.

void CPPProcess::sigmaKin() {
  // Set the parameters which change event by event
  pars->setDependentParameters();
  pars->setDependentCouplings();
  static bool firsttime = true;
  if (firsttime && m_verbose) {
    pars->printDependentParameters();
    pars->printDependentCouplings();
    firsttime = false;
  }

  // Reset color flows
  for (int i = 0; i < 1; i++)
    jamp2[0][i] = 0.;

  // Local variables and constants
  const int ncomb = 16;
  static bool goodhel[ncomb] = {ncomb * false};
  static int ntry = 0, sum_hel = 0, ngood = 0;
  static int igood[ncomb];
  static int jhel;
  // thrust::complex<double> **wfs;
  double t[nprocesses];
  // Denominators: spins, colors and identical particles
  const int denominators[nprocesses] = {4};

  ntry = ntry + 1;

  // Reset the matrix elements
  for (int i = 0; i < nprocesses; i++) {
    matrix_element[i] = 0.;
  }

  if (sum_hel == 0 || ntry < 10) {
    // Calculate the matrix element for all helicities
    for (int ihel = 0; ihel < ncomb; ihel++) {
      if (goodhel[ihel] || ntry < 2) {

        call_wavefunctions_kernel(ihel);
        t[0] = matrix_1_epem_mupmum();

        double tsum = 0;
        for (int iproc = 0; iproc < nprocesses; iproc++) {
          matrix_element[iproc] += t[iproc];
          tsum += t[iproc];
        }
        // Store which helicities give non-zero result
        if (tsum != 0. && !goodhel[ihel]) {
          goodhel[ihel] = true;
          ngood++;
          igood[ngood] = ihel;
        }
      }
    }
    jhel = 0;
    sum_hel = min(sum_hel, ngood);
  } else {
    // Only use the "good" helicities
    for (int j = 0; j < sum_hel; j++) {
      jhel++;
      if (jhel >= ngood)
        jhel = 0;
      double hwgt = double(ngood) / double(sum_hel);
      int ihel = igood[jhel];

      call_wavefunctions_kernel(ihel);
      t[0] = matrix_1_epem_mupmum();

      for (int iproc = 0; iproc < nprocesses; iproc++) {
        matrix_element[iproc] += t[iproc] * hwgt;
      }
    }
  }

  for (int i = 0; i < nprocesses; i++)
    matrix_element[i] /= denominators[i];
}

//--------------------------------------------------------------------------
// Evaluate |M|^2, including incoming flavour dependence.

double CPPProcess::sigmaHat() {
  // Select between the different processes
  if (id1 == -11 && id2 == 11) {
    // Add matrix elements for processes with beams (-11, 11)
    return matrix_element[0];
  } else {
    // Return 0 if not correct initial state assignment
    return 0.;
  }
}

//==========================================================================
// Private class member functions

//--------------------------------------------------------------------------
// Evaluate |M|^2 for each subprocess

void CPPProcess::call_wavefunctions_kernel(int ihel) {

  m_timer.Start();

  gMG5_sm::calculate_wavefunctions<<<1, 1>>>(
      m->tperm, m->thelicities[ihel], m->mME, m->tp, m->tamp, m->tw, pars->GC_3,
      pars->GC_51, pars->GC_59, pars->mdl_MZ, pars->mdl_WZ);
  hipDeviceSynchronize();

  gpuErrchk(hipMemcpy((void *)amp, (void *)m->tamp,
                       m->tnamplitudes * sizeof(thrust::complex<double>),
                       hipMemcpyDeviceToHost));

  float gputime = m_timer.GetDuration();
  std::cout << "Wave function time: " << gputime << std::endl;
}

/*
void CPPProcess::calculate_wavefunctions(const int perm[], const int hel[]) {
  // Calculate wavefunctions for all processes
  // int i, j;


  // Calculate all wavefunctions
  gMG5_sm::oxxxxx<<<1, 1>>>(p[perm[0]], mME[0], hel[0], -1, 0);
  gMG5_sm::ixxxxx<<<1, 1>>>(p[perm[1]], mME[1], hel[1], +1, 1);
  gMG5_sm::ixxxxx<<<1, 1>>>(p[perm[2]], mME[2], hel[2], -1, 2);
  gMG5_sm::oxxxxx<<<1, 1>>>(p[perm[3]], mME[3], hel[3], +1, 3);

  gMG5_sm::FFV1P0_3<<<1, 1>>>(1, 0, pars->GC_3, pars->ZERO, pars->ZERO, 4);
  gMG5_sm::FFV2_4_3<<<1, 1>>>(1, 0, -pars->GC_51, pars->GC_59, pars->mdl_MZ,
                              pars->mdl_WZ, 5);
  // Calculate all amplitudes
  // Amplitude(s) for diagram number 0
  gMG5_sm::FFV1_0<<<1, 1>>>(2, 3, 4, pars->GC_3, &m->tamp[0]);
  gMG5_sm::FFV2_4_0<<<1, 1>>>(2, 3, 5, -pars->GC_51, pars->GC_59, &m->tamp[1]);
  hipDeviceSynchronize();

}
*/

double CPPProcess::matrix_1_epem_mupmum() {
  int i, j;
  // Local variables
  // const int ngraphs = 2;
  const int ncolor = 1;
  std::complex<double> ztemp;
  std::complex<double> jamp[ncolor];
  // The color matrix;
  static const double denom[ncolor] = {1};
  static const double cf[ncolor][ncolor] = {{1}};

  // Calculate color flows
  jamp[0] = -m->tamp[0] - m->tamp[1];

  // Sum and square the color flows to get the matrix element
  double matrix = 0;
  for (i = 0; i < ncolor; i++) {
    ztemp = 0.;
    for (j = 0; j < ncolor; j++)
      ztemp = ztemp + cf[i][j] * jamp[j];
    matrix = matrix + real(ztemp * conj(jamp[i])) / denom[i];
  }

  // Store the leading color flows for choice of color
  for (i = 0; i < ncolor; i++)
    jamp2[0][i] += real(jamp[i] * conj(jamp[i]));

  return matrix;
}
