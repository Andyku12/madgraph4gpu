#include "hip/hip_runtime.h"
#include <algorithm> // perf stats
#include <cstring>
#include <iomanip>
#include <iostream>
#include <numeric> // perf stats
#include <unistd.h>
#include <vector>

#include "CPPProcess.h"
//#include "HelAmps_sm.h"

#include "vrambo.h"
#include "timer.h"

#define TIMERTYPE std::chrono::high_resolution_clock

bool is_number(const char *s) {
  const char *t = s;
  while (*t != '\0' && isdigit(*t))
    ++t;
  return strlen(s) == t - s;
}

int usage(char* argv0, int ret = 1) {
  std::cout << "Usage: " << argv0 
            << " [--verbose|-v] [--debug|-d] [--performance|-p]"
            << " [#gpuBlocksPerGrid #gpuThreadsPerBlock] #iterations" << std::endl;
  return ret;
}

int main(int argc, char **argv) {
  bool verbose = false, debug = false, perf = false;
  int niter = 0;
  int gpublocks = 1;
  int gputhreads = 1;
  std::vector<int> numvec;
  Timer<TIMERTYPE> timer;
  std::vector<float> wavetimes;


  for (int argn = 1; argn < argc; ++argn) {
    if (strcmp(argv[argn], "--verbose") == 0 || strcmp(argv[argn], "-v") == 0)
      verbose = true;
    else if (strcmp(argv[argn], "--debug") == 0 ||
             strcmp(argv[argn], "-d") == 0)
      debug = true;
    else if (strcmp(argv[argn], "--performance") == 0 ||
             strcmp(argv[argn], "-p") == 0)
      perf = true;
    else if (is_number(argv[argn]))
      numvec.push_back(atoi(argv[argn]));
    else
      return usage(argv[0]);
  }
  int veclen = numvec.size();
  if (veclen == 3) {
    gpublocks = numvec[0];
    gputhreads = numvec[1];
    niter = numvec[2];
  } else if (veclen == 1) {
    niter = numvec[0];
  } else {
    return usage(argv[0]);
  }

  if (niter == 0)
    return usage(argv[0]);

  //std::cout << "Calling hipFree... " << std::endl;
  gpuErrchk3( hipFree( 0 ) ); // SLOW!
  //std::cout << "Calling hipFree... done" << std::endl;

  if (verbose)
    std::cout << "# iterations: " << niter << std::endl;

  // Create a process object
  CPPProcess process(niter, gpublocks, gputhreads, verbose, debug);

  // Read param_card and set parameters
  process.initProc("../../Cards/param_card.dat");

  const double energy = 1500;

  const int meGeVexponent = -(2 * process.nexternal - 8);

  // Memory structures for input momenta and output matrix elements on host and device
  const int ndim = gpublocks * gputhreads;
  const int npar = process.nexternal; // for this process (eemumu): npar=4
  const int np4 = 4; // dimension of 4-momenta (E,px,py,pz): copy all of them from rambo

  double* hstMomenta = new double[npar*np4*ndim]; // SOA[npar][np4][ndim] (previously was: lp)
  int nbytesMomenta = np4*npar*ndim * sizeof(double);
  double* devMomenta = 0; // (previously was: allMomenta)
  gpuErrchk3( hipMalloc( &devMomenta, nbytesMomenta ) );

  double* hstMEs = new double[ndim]; // (previously was: meHostPtr)
  int nbytesMEs = ndim * sizeof(double);
  double* devMEs = 0; // (previously was: meDevPtr)
  gpuErrchk3( hipMalloc( &devMEs, nbytesMEs ) );

  std::vector<double> matrixelementvector;

  for (int iiter = 0; iiter < niter; ++iiter) {

    //std::cout << "Iteration #" << iiter+1 << " of " << niter << std::endl;
    // Get a vector of ndim phase space points
    double weight; // dummy in this test application
    std::vector<std::vector<double *>> rmbMomenta = // AOS[ndim][npar][np4] (previously was: p)
      get_momenta(process.ninitial, energy, process.getMasses(), weight, ndim); // SLOW!
    //std::cout << "Got momenta" << std::endl;

    // Set momenta for this event by copying them from the rambo output
    for (int idim = 0; idim < ndim; ++idim)
      for (int ipar = 0; ipar < npar; ++ipar)
        for (int ip4 = 0; ip4 < np4; ++ip4)
          hstMomenta[ipar*ndim*np4 + ip4*ndim + idim] = // SOA[npar][np4][ndim]
            rmbMomenta[idim][ipar][ip4];
    gpuErrchk3( hipMemcpy( devMomenta, hstMomenta, nbytesMomenta, hipMemcpyHostToDevice ) );

   //process.preSigmaKin();

    if (perf) {
      timer.Start();
    }

    // Evaluate matrix element
    // later process.sigmaKin(ncomb, goodhel, ntry, sum_hel, ngood, igood,
    // jhel);
    sigmaKin<<<gpublocks, gputhreads>>>(devMomenta,  devMEs);//, debug, verbose);
    gpuErrchk3( hipPeekAtLastError() );

    gpuErrchk3( hipMemcpy( hstMEs, devMEs, nbytesMEs, hipMemcpyDeviceToHost ) );

    if (verbose)
      std::cout << "***********************************" << std::endl
                << "Iteration #" << iiter+1 << " of " << niter << std::endl;

    if (perf) {
      float gputime = timer.GetDuration();
      wavetimes.push_back(gputime);
      if (verbose)
        std::cout << "Wave function time: " << gputime << std::endl;
    }

    if (verbose || perf) {

      for (int idim = 0; idim < ndim; ++idim) {

        if (verbose) {
          std::cout << "Momenta:" << std::endl;
          for (int ipar = 0; ipar < npar; ipar++)
            std::cout << std::setw(4) << ipar + 1
                      << setiosflags(std::ios::scientific)
                      << std::setw(14) << rmbMomenta[idim][ipar][0]
                      << setiosflags(std::ios::scientific)
                      << std::setw(14) << rmbMomenta[idim][ipar][1]
                      << setiosflags(std::ios::scientific)
                      << std::setw(14) << rmbMomenta[idim][ipar][2]
                      << setiosflags(std::ios::scientific)
                      << std::setw(14) << rmbMomenta[idim][ipar][3] << std::endl;
          std::cout << std::string(80, '-') << std::endl;
        }

        // Display matrix elements
        for (int iproc = 0; iproc < process.nprocesses; iproc++) {
          if (verbose)
            std::cout << " Matrix element = "
                      //	 << setiosflags(ios::fixed) << setprecision(17)
                      << hstMEs[iproc*1 + idim] << " GeV^" << meGeVexponent << std::endl;
          if (perf)
            matrixelementvector.push_back(hstMEs[iproc*1 + idim]);
        }
        if (verbose)
          std::cout << std::string(80, '-') << std::endl;
      }
    } else if (!debug) {
      std::cout << ".";
    }

    for (std::vector<std::vector<double *>>::iterator it = rmbMomenta.begin();
         it != rmbMomenta.end(); ++it) {
      for (std::vector<double *>::iterator jt = it->begin(); jt != it->end();
           ++jt) {
        delete[] & (**jt);
      }
    }
  }

  if (!(verbose || debug || perf)) {
    std::cout << std::endl;
  }

  if (perf) {
    float sum = std::accumulate(wavetimes.begin(), wavetimes.end(), 0.0);
    int num_wts = wavetimes.size();
    float mean = sum / num_wts;
    float sq_sum = std::inner_product(wavetimes.begin(), wavetimes.end(),
                                      wavetimes.begin(), 0.0);
    float stdev = std::sqrt(sq_sum / num_wts - mean * mean);
    std::vector<float>::iterator mintime =
        std::min_element(wavetimes.begin(), wavetimes.end());
    std::vector<float>::iterator maxtime =
        std::max_element(wavetimes.begin(), wavetimes.end());

    int num_mes = matrixelementvector.size();
    float sumelem = std::accumulate(matrixelementvector.begin(), matrixelementvector.end(), 0.0);
    float meanelem = sumelem / num_mes;
    float sqselem = std::inner_product(matrixelementvector.begin(), matrixelementvector.end(), 
                                       matrixelementvector.begin(), 0.0);
    float stdelem = std::sqrt(sqselem / num_mes - meanelem * meanelem);
    std::vector<double>::iterator maxelem = std::max_element(
        matrixelementvector.begin(), matrixelementvector.end());
    std::vector<double>::iterator minelem = std::min_element(
        matrixelementvector.begin(), matrixelementvector.end());

    std::cout << "***********************************" << std::endl
              << "NumIterations         = " << niter << std::endl
              << "NumThreadsPerBlock    = " << gputhreads << std::endl
              << "NumBlocksPerGrid      = " << gpublocks << std::endl
              << "-----------------------------------" << std::endl
              << "NumberOfEntries       = " << num_wts << std::endl
              << std::scientific
              << "TotalTimeInWaveFuncs  = " << sum << " sec" << std::endl
              << "MeanTimeInWaveFuncs   = " << mean << " sec" << std::endl
              << "StdDevTimeInWaveFuncs = " << stdev << " sec" << std::endl
              << "MinTimeInWaveFuncs    = " << *mintime << " sec" << std::endl
              << "MaxTimeInWaveFuncs    = " << *maxtime << " sec" << std::endl
              << "-----------------------------------" << std::endl
              << "ProcessID:            = " << getpid() << std::endl
              << "NProcesses            = " << process.nprocesses << std::endl
              << "NumMatrixElements     = " << num_mes << std::endl
              << "MatrixElementsPerSec  = " << num_mes/sum << " sec^-1" << std::endl;

    std::cout << "***********************************" << std::endl
              << "NumMatrixElements     = " << num_mes << std::endl
              << std::scientific
              << "MeanMatrixElemValue   = " << meanelem << " GeV^" << meGeVexponent << std::endl
              << "StdErrMatrixElemValue = " << stdelem/sqrt(num_mes) << " GeV^" << meGeVexponent << std::endl
              << "StdDevMatrixElemValue = " << stdelem << " GeV^" << meGeVexponent << std::endl
              << "MinMatrixElemValue    = " << *minelem << " GeV^" << meGeVexponent << std::endl
              << "MaxMatrixElemValue    = " << *maxelem << " GeV^" << meGeVexponent << std::endl;
  }

  delete[] hstMomenta;
  gpuErrchk3( hipFree( devMEs ) );
  gpuErrchk3( hipFree( devMomenta ) );
  gpuErrchk3( hipDeviceReset() ); // this is needed by cuda-memcheck --leak-check full
}
