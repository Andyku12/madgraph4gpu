#include "hip/hip_runtime.h"
#include <algorithm> // perf stats
#include <cstring>
#include <iomanip>
#include <iostream>
#include <numeric> // perf stats
#include <unistd.h>
#include <vector>

#include "mgOnGpuConfig.h"
#include "vrambo.h"

#include "CPPProcess.h"
#include "timermap.h"

bool is_number(const char *s) {
  const char *t = s;
  while (*t != '\0' && isdigit(*t))
    ++t;
  return strlen(s) == t - s;
}

int usage(char* argv0, int ret = 1) {
  std::cout << "Usage: " << argv0
            << " [--verbose|-v] [--debug|-d] [--performance|-p]"
            << " [#gpuBlocksPerGrid #gpuThreadsPerBlock] #iterations" << std::endl;
  return ret;
}

int main(int argc, char **argv)
{
  // READ COMMAND LINE ARGUMENTS
  bool verbose = false;
  bool debug = false;
  bool perf = false;
  int niter = 0;
  int gpublocks = 1;
  int gputhreads = 1;
  std::vector<int> numvec;

  for (int argn = 1; argn < argc; ++argn) {
    if (strcmp(argv[argn], "--verbose") == 0 || strcmp(argv[argn], "-v") == 0)
      verbose = true;
    else if (strcmp(argv[argn], "--debug") == 0 ||
             strcmp(argv[argn], "-d") == 0)
      debug = true;
    else if (strcmp(argv[argn], "--performance") == 0 ||
             strcmp(argv[argn], "-p") == 0)
      perf = true;
    else if (is_number(argv[argn]))
      numvec.push_back(atoi(argv[argn]));
    else
      return usage(argv[0]);
  }

  int veclen = numvec.size();
  if (veclen == 3) {
    gpublocks = numvec[0];
    gputhreads = numvec[1];
    niter = numvec[2];
  } else if (veclen == 1) {
    niter = numvec[0];
  } else {
    return usage(argv[0]);
  }

  if (niter == 0)
    return usage(argv[0]);

  const int ndim = gpublocks * gputhreads; // number of events (threads) in one iteration
#if defined MGONGPU_LAYOUT_ASA
  using mgOnGpu::nepp;
  if ( gputhreads%nepp != 0 )
  {
    std::cout << "ERROR! #threads/block should be a multiple of " << nepp << std::endl;
    return usage(argv[0]);
  }
  const int npag = ndim/nepp; // number of ASA pages needed for ndim events
#endif

  if (verbose)
    std::cout << "# iterations: " << niter << std::endl;

  // *** START THE NEW TIMERS ***
  mgOnGpu::TimerMap timermap;

  // === STEP 0 - INITIALISE

  // --- 00. Initialise cuda (call hipFree to ease cuda profile analysis)
  const std::string cdfrKey = "00 CudaFree";
  timermap.start( cdfrKey );

  //std::cout << "Calling hipFree... " << std::endl;
  gpuErrchk3( hipFree( 0 ) ); // SLOW!
  //std::cout << "Calling hipFree... done" << std::endl;

  // --- 0a. Initialise physics process
  const std::string procKey = "0a InitProc";
  timermap.start( procKey );

  // Create a process object
  CPPProcess process(niter, gpublocks, gputhreads, verbose, debug);

  // Read param_card and set parameters
  process.initProc("../../Cards/param_card.dat");

  const double energy = 1500;
  const int meGeVexponent = -(2 * process.nexternal - 8);

  // --- 0b. Allocate memory structures
  const std::string alloKey = "0b MemAlloc";
  timermap.start( alloKey );

  // Memory structures for input momenta and output matrix elements on host and device
  const int npar = process.nexternal; // for this process (eemumu): npar=4 (e+, e-, mu+, mu-)
  const int nparf = npar - process.ninitial; // for this process (eemumu): nparf=2 (mu+, mu-)
  const int np4 = 4; // dimension of 4-momenta (E,px,py,pz): copy all of them from rambo

#if defined MGONGPU_LAYOUT_ASA
  double* rnarray = new double[npag*nparf*np4*nepp]; // AOSOA[npag][nparf][np4][nepp] (NB: ndim=npag*nepp)
#elif defined MGONGPU_LAYOUT_SOA
  double* rnarray = new double[nparf*np4*ndim]; // SOA[npar][np4][ndim]
#elif defined MGONGPU_LAYOUT_AOS
  double* rnarray = new double[nparf*np4*ndim]; // AOS[ndim][npar][np4]
#endif

  const int nbytesMomenta = np4*npar*ndim * sizeof(double); // (NB: ndim=npag*nepp for ASA layouts)
#if defined MGONGPU_LAYOUT_ASA
  double* hstMomenta = 0; // AOSOA[npag][npar][np4][nepp] (previously was: lp)
#elif defined MGONGPU_LAYOUT_SOA
  double* hstMomenta = 0; // SOA[npar][np4][ndim] (previously was: lp)
#elif defined MGONGPU_LAYOUT_AOS
  double* hstMomenta = 0; // AOS[ndim][npar][np4] (previously was: lp)
#endif
  gpuErrchk3( hipHostMalloc( &hstMomenta, nbytesMomenta ) );
  double* devMomenta = 0; // (previously was: allMomenta)
  gpuErrchk3( hipMalloc( &devMomenta, nbytesMomenta ) );

  const int nbytesMEs = ndim * sizeof(double); //  (NB: ndim=npag*nepp for ASA layouts)
  double* hstMEs = 0; // (previously was: meHostPtr)
  gpuErrchk3( hipHostMalloc( &hstMEs, nbytesMEs ) );
  double* devMEs = 0; // (previously was: meDevPtr)
  gpuErrchk3( hipMalloc( &devMEs, nbytesMEs ) );

  double masses[npar];
  for (int ipar = 0; ipar < npar; ++ipar) // loop over nexternal particles
    masses[ipar] = process.getMasses()[ipar];

  std::vector<float> wavetimes;
  std::vector<double> matrixelementvector;

  // **************************************
  // *** START MAIN LOOP ON #ITERATIONS ***
  // **************************************

  for (int iiter = 0; iiter < niter; ++iiter)
  {
    //std::cout << "Iteration #" << iiter+1 << " of " << niter << std::endl;

    // === STEP 1 OF 3
    // Generate all relevant numbers to build ndim events (i.e. ndim phase space points)
    const std::string rngnKey = "1  RnNumGen";
    timermap.start( rngnKey );
    generateRnArray( rnarray, nparf, ndim );
    //std::cout << "Got random numbers" << std::endl;

    // === STEP 2 OF 3
    // Map random numbers to particle momenta for each of ndim events
    const std::string rambKey = "2  RamboMap";
    timermap.start( rambKey );
    double weights[ndim]; // dummy in this test application
    get_momenta( process.ninitial, energy, masses, rnarray, hstMomenta, weights, npar, ndim );
    //std::cout << "Got momenta" << std::endl;

    // === STEP 3 OF 3
    // Evaluate matrix elements for all ndim events
    // 3a. Copy momenta from host to device
    // 3b. Evaluate MEs on the device
    // 3c. Copy MEs back from device to host

    // --- 3a. CopyHToD
    const std::string htodKey = "3a CopyHToD";
    timermap.start( htodKey );
    gpuErrchk3( hipMemcpy( devMomenta, hstMomenta, nbytesMomenta, hipMemcpyHostToDevice ) );

    // *** START THE OLD TIMER ***
    float gputime = 0;

    // --- 3b. SigmaKin
    const std::string skinKey = "3b SigmaKin";
    timermap.start( skinKey );

    sigmaKin<<<gpublocks, gputhreads>>>(devMomenta,  devMEs);//, debug, verbose);
    gpuErrchk3( hipPeekAtLastError() );

    // --- 3c. CopyDToH
    const std::string dtohKey = "3c CopyDToH";
    gputime += timermap.start( dtohKey );

    gpuErrchk3( hipMemcpy( hstMEs, devMEs, nbytesMEs, hipMemcpyDeviceToHost ) );

    // === STEP 9 FINALISE
    // --- 9a Dump within the loop
    // *** STOP THE OLD TIMER ***
    const std::string loopKey = "9a DumpLoop";
    gputime += timermap.start(loopKey);
    wavetimes.push_back( gputime );

    if (verbose)
    {
      std::cout << "***********************************" << std::endl
                << "Iteration #" << iiter+1 << " of " << niter << std::endl;
      if (perf) std::cout << "Wave function time: " << gputime << std::endl;
    }

    if (verbose || perf)
    {
      for (int idim = 0; idim < ndim; ++idim) // Loop over all events in this iteration
      {
#if defined MGONGPU_LAYOUT_ASA
        const int ipag = idim/nepp; // #eventpage in this iteration
        const int iepp = idim%nepp; // #event in the current eventpage in this iteration
#endif
        if (verbose)
        {
          std::cout << "Momenta:" << std::endl;
          for (int ipar = 0; ipar < npar; ipar++)
          {
#if defined MGONGPU_LAYOUT_ASA
            std::cout << std::setw(4) << ipar + 1
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipag*npar*np4*nepp + ipar*nepp*np4 + 0*nepp + iepp] // AOSOA[ipag][ipar][0][iepp]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipag*npar*np4*nepp + ipar*nepp*np4 + 1*nepp + iepp] // AOSOA[ipag][ipar][1][iepp]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipag*npar*np4*nepp + ipar*nepp*np4 + 2*nepp + iepp] // AOSOA[ipag][ipar][2][iepp]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipag*npar*np4*nepp + ipar*nepp*np4 + 3*nepp + iepp] // AOSOA[ipag][ipar][3][iepp]
                      << std::endl;
#elif defined MGONGPU_LAYOUT_SOA
            std::cout << std::setw(4) << ipar + 1
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipar*ndim*np4 + 0*ndim + idim] // SOA[ipar][0][idim]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipar*ndim*np4 + 1*ndim + idim] // SOA[ipar][1][idim]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipar*ndim*np4 + 2*ndim + idim] // SOA[ipar][2][idim]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[ipar*ndim*np4 + 3*ndim + idim] // SOA[ipar][3][idim]
                      << std::endl;
#elif defined MGONGPU_LAYOUT_AOS
            std::cout << std::setw(4) << ipar + 1
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[idim*npar*np4 + ipar*np4 + 0] // AOS[idim][ipar][0]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[idim*npar*np4 + ipar*np4 + 1] // AOS[idim][ipar][1]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[idim*npar*np4 + ipar*np4 + 2] // AOS[idim][ipar][2]
                      << setiosflags(std::ios::scientific) << std::setw(14)
                      << hstMomenta[idim*npar*np4 + ipar*np4 + 3] // AOS[idim][ipar][3]
                      << std::endl;
#endif
          }
          std::cout << std::string(80, '-') << std::endl;
        }

        // Display matrix elements
        for (int iproc = 0; iproc < process.nprocesses; iproc++) {
          if (verbose)
            std::cout << " Matrix element = "
              //   << setiosflags(ios::fixed) << setprecision(17)
                      << hstMEs[iproc*1 + idim] << " GeV^" << meGeVexponent << std::endl;
          if (perf)
            matrixelementvector.push_back(hstMEs[iproc*1 + idim]);
        }
        if (verbose)
          std::cout << std::string(80, '-') << std::endl;
      }
    }
    else if (!debug)
    {
      std::cout << ".";
    }
  }

  // **************************************
  // *** END MAIN LOOP ON #ITERATIONS ***
  // **************************************

  // === STEP 9 FINALISE
  // --- 9b Dump after the loop
  const std::string dumpKey = "9b DumpAll ";
  timermap.start(dumpKey);

  if (!(verbose || debug || perf))
  {
    std::cout << std::endl;
  }

  if (perf)
  {
    float sum = std::accumulate(wavetimes.begin(), wavetimes.end(), 0.0);
    int num_wts = wavetimes.size();
    float mean = sum / num_wts;
    float sq_sum = std::inner_product(wavetimes.begin(), wavetimes.end(),
                                      wavetimes.begin(), 0.0);
    float stdev = std::sqrt(sq_sum / num_wts - mean * mean);
    std::vector<float>::iterator mintime =
      std::min_element(wavetimes.begin(), wavetimes.end());
    std::vector<float>::iterator maxtime =
      std::max_element(wavetimes.begin(), wavetimes.end());

    int num_mes = matrixelementvector.size();
    float sumelem = std::accumulate(matrixelementvector.begin(), matrixelementvector.end(), 0.0);
    float meanelem = sumelem / num_mes;
    float sqselem = std::inner_product(matrixelementvector.begin(), matrixelementvector.end(),
                                       matrixelementvector.begin(), 0.0);
    float stdelem = std::sqrt(sqselem / num_mes - meanelem * meanelem);
    std::vector<double>::iterator maxelem = std::max_element(
                                                             matrixelementvector.begin(), matrixelementvector.end());
    std::vector<double>::iterator minelem = std::min_element(
                                                             matrixelementvector.begin(), matrixelementvector.end());

    std::cout << "***********************************" << std::endl
              << "NumIterations         = " << niter << std::endl
              << "NumThreadsPerBlock    = " << gputhreads << std::endl
              << "NumBlocksPerGrid      = " << gpublocks << std::endl
              << "-----------------------------------" << std::endl
#if defined MGONGPU_LAYOUT_ASA
              << "Memory layout         = AOSOA " << std::endl
#elif defined MGONGPU_LAYOUT_SOA
              << "Memory layout         = SOA " << std::endl
#elif defined MGONGPU_LAYOUT_AOS
              << "Memory layout         = AOS " << std::endl
#endif
              << "-----------------------------------" << std::endl
              << "NumberOfEntries       = " << num_wts << std::endl
              << std::scientific
              << "TotalTimeInWaveFuncs  = " << sum << " sec" << std::endl
              << "MeanTimeInWaveFuncs   = " << mean << " sec" << std::endl
              << "StdDevTimeInWaveFuncs = " << stdev << " sec" << std::endl
              << "MinTimeInWaveFuncs    = " << *mintime << " sec" << std::endl
              << "MaxTimeInWaveFuncs    = " << *maxtime << " sec" << std::endl
              << "-----------------------------------" << std::endl
              << "ProcessID:            = " << getpid() << std::endl
              << "NProcesses            = " << process.nprocesses << std::endl
              << "NumMatrixElements     = " << num_mes << std::endl
              << "MatrixElementsPerSec  = " << num_mes/sum << " sec^-1" << std::endl;

    std::cout << "***********************************" << std::endl
              << "NumMatrixElements     = " << num_mes << std::endl
              << std::scientific
              << "MeanMatrixElemValue   = " << meanelem << " GeV^" << meGeVexponent << std::endl
              << "StdErrMatrixElemValue = " << stdelem/sqrt(num_mes) << " GeV^" << meGeVexponent << std::endl
              << "StdDevMatrixElemValue = " << stdelem << " GeV^" << meGeVexponent << std::endl
              << "MinMatrixElemValue    = " << *minelem << " GeV^" << meGeVexponent << std::endl
              << "MaxMatrixElemValue    = " << *maxelem << " GeV^" << meGeVexponent << std::endl;
  }

  // --- 9c Free memory structures
  const std::string freeKey = "9c MemFree ";
  timermap.start( freeKey );

  delete[] rnarray;

  //delete[] hstMEs;
  //delete[] hstMomenta;
  gpuErrchk3( hipHostFree( hstMEs ) );
  gpuErrchk3( hipHostFree( hstMomenta ) );

  gpuErrchk3( hipFree( devMEs ) );
  gpuErrchk3( hipFree( devMomenta ) );

  gpuErrchk3( hipDeviceReset() ); // this is needed by cuda-memcheck --leak-check full

  // *** STOP THE NEW TIMERS ***
  timermap.stop();
  if (perf)
  {
    std::cout << "***********************************" << std::endl;
    timermap.dump();
    std::cout << "***********************************" << std::endl;
  }
}
