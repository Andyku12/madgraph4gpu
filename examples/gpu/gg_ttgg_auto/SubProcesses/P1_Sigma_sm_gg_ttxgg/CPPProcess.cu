#include "hip/hip_runtime.h"
//==========================================================================
// This file has been automatically generated for C++ Standalone by
// MadGraph5_aMC@NLO v. 2.7.3.py3, 2020-06-28
// By the MadGraph5_aMC@NLO Development Team
// Visit launchpad.net/madgraph5 and amcatnlo.web.cern.ch
//==========================================================================

#include "CPPProcess.h"
#include "HelAmps_sm.h"

#include <algorithm> 
#include <iostream> 
#include <thrust/complex.h> 

using namespace MG5_sm; 

//==========================================================================
// Class member functions for calculating the matrix elements for
// Process: g g > t t~ g g WEIGHTED<=4 @1

__constant__ int cHel[64][6]; 
// __constant__ double cmME[6]; value hardcoded now
// extern __constant__ int cPerm[4];
// 
__constant__ double cIPC[6];  // coupling ?
__constant__ double cIPD[2]; 


// Evaluate |M|^2 for each subprocess

__device__ void calculate_wavefunctions(int ihel, double local_mom[6][3],
    double &matrix)
{
  thrust::complex<double> amp[159]; 
  // Calculate wavefunctions for all processes
  thrust::complex<double> w[26][6]; 
  vxxxxx(local_mom[0], 0., cHel[ihel][0], -1, w[0]); 
  vxxxxx(local_mom[1], 0., cHel[ihel][1], -1, w[1]); 
  oxxxxx(local_mom[2], cIPD[0], cHel[ihel][2], +1, w[2]); 
  ixxxxx(local_mom[3], cIPD[0], cHel[ihel][3], -1, w[3]); 
  vxxxxx(local_mom[4], 0., cHel[ihel][4], +1, w[4]); 
  vxxxxx(local_mom[5], 0., cHel[ihel][5], +1, w[5]); 
  VVV1P0_1(w[0], w[1], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[6]);
  FFV1P0_3(w[3], w[2], thrust::complex<double> (cIPC[2], cIPC[3]), 0., 0.,
      w[7]);
  // Amplitude(s) for diagram number 1
  VVVV1_0(w[6], w[7], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[0]);
  VVVV3_0(w[6], w[7], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[1]);
  VVVV4_0(w[6], w[7], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[2]);
  VVV1P0_1(w[6], w[4], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[8]);
  // Amplitude(s) for diagram number 2
  VVV1_0(w[7], w[5], w[8], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[3]);
  VVV1P0_1(w[6], w[5], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[9]);
  // Amplitude(s) for diagram number 3
  VVV1_0(w[7], w[4], w[9], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[4]);
  VVV1P0_1(w[4], w[5], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[10]);
  // Amplitude(s) for diagram number 4
  VVV1_0(w[6], w[7], w[10], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[5]);
  FFV1_1(w[2], w[4], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[11]);
  FFV1_2(w[3], w[6], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[12]);
  // Amplitude(s) for diagram number 5
  FFV1_0(w[12], w[11], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[6]);
  // Amplitude(s) for diagram number 6
  FFV1_0(w[3], w[11], w[9], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[7]);
  FFV1_2(w[3], w[5], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[13]);
  // Amplitude(s) for diagram number 7
  FFV1_0(w[13], w[11], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[8]);
  FFV1_1(w[2], w[5], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[14]);
  // Amplitude(s) for diagram number 8
  FFV1_0(w[12], w[14], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[9]);
  // Amplitude(s) for diagram number 9
  FFV1_0(w[3], w[14], w[8], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[10]);
  FFV1_2(w[3], w[4], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[15]);
  // Amplitude(s) for diagram number 10
  FFV1_0(w[15], w[14], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[11]);
  FFV1_1(w[2], w[6], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[16]);
  // Amplitude(s) for diagram number 11
  FFV1_0(w[15], w[16], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[12]);
  // Amplitude(s) for diagram number 12
  FFV1_0(w[15], w[2], w[9], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[13]);
  // Amplitude(s) for diagram number 13
  FFV1_0(w[13], w[16], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[14]);
  // Amplitude(s) for diagram number 14
  FFV1_0(w[13], w[2], w[8], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[15]);
  // Amplitude(s) for diagram number 15
  FFV1_0(w[3], w[16], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[16]);
  // Amplitude(s) for diagram number 16
  FFV1_0(w[12], w[2], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[17]);
  FFV1_1(w[2], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[12]);
  FFV1_2(w[3], w[1], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[16]);
  FFV1_1(w[12], w[4], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[8]);
  // Amplitude(s) for diagram number 17
  FFV1_0(w[16], w[8], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[18]);
  FFV1_1(w[12], w[5], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[9]);
  // Amplitude(s) for diagram number 18
  FFV1_0(w[16], w[9], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[19]);
  // Amplitude(s) for diagram number 19
  FFV1_0(w[16], w[12], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[20]);
  VVV1P0_1(w[1], w[4], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[6]);
  FFV1P0_3(w[3], w[12], thrust::complex<double> (cIPC[2], cIPC[3]), 0., 0.,
      w[17]);
  // Amplitude(s) for diagram number 20
  VVV1_0(w[6], w[5], w[17], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[21]);
  // Amplitude(s) for diagram number 21
  FFV1_0(w[3], w[9], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[22]);
  // Amplitude(s) for diagram number 22
  FFV1_0(w[13], w[12], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[23]);
  VVV1P0_1(w[1], w[5], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[18]);
  // Amplitude(s) for diagram number 23
  VVV1_0(w[18], w[4], w[17], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[24]);
  // Amplitude(s) for diagram number 24
  FFV1_0(w[3], w[8], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[25]);
  // Amplitude(s) for diagram number 25
  FFV1_0(w[15], w[12], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[26]);
  FFV1_1(w[12], w[1], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[19]);
  // Amplitude(s) for diagram number 26
  FFV1_0(w[15], w[19], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[27]);
  // Amplitude(s) for diagram number 27
  FFV1_0(w[15], w[9], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[28]);
  // Amplitude(s) for diagram number 28
  FFV1_0(w[13], w[19], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[29]);
  // Amplitude(s) for diagram number 29
  FFV1_0(w[13], w[8], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[30]);
  // Amplitude(s) for diagram number 30
  FFV1_0(w[3], w[19], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[31]);
  // Amplitude(s) for diagram number 31
  VVV1_0(w[1], w[10], w[17], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[32]);
  VVVV1P0_1(w[1], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[17]);
  VVVV3P0_1(w[1], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[19]);
  VVVV4P0_1(w[1], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[8]);
  // Amplitude(s) for diagram number 32
  FFV1_0(w[3], w[12], w[17], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[33]);
  FFV1_0(w[3], w[12], w[19], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[34]);
  FFV1_0(w[3], w[12], w[8], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[35]);
  FFV1_2(w[3], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[12]);
  FFV1_1(w[2], w[1], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[9]);
  FFV1_2(w[12], w[4], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[20]);
  // Amplitude(s) for diagram number 33
  FFV1_0(w[20], w[9], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[36]);
  FFV1_2(w[12], w[5], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[21]);
  // Amplitude(s) for diagram number 34
  FFV1_0(w[21], w[9], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[37]);
  // Amplitude(s) for diagram number 35
  FFV1_0(w[12], w[9], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[38]);
  FFV1P0_3(w[12], w[2], thrust::complex<double> (cIPC[2], cIPC[3]), 0., 0.,
      w[22]);
  // Amplitude(s) for diagram number 36
  VVV1_0(w[6], w[5], w[22], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[39]);
  // Amplitude(s) for diagram number 37
  FFV1_0(w[21], w[2], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[40]);
  // Amplitude(s) for diagram number 38
  FFV1_0(w[12], w[14], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[41]);
  // Amplitude(s) for diagram number 39
  VVV1_0(w[18], w[4], w[22], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[42]);
  // Amplitude(s) for diagram number 40
  FFV1_0(w[20], w[2], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[43]);
  // Amplitude(s) for diagram number 41
  FFV1_0(w[12], w[11], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[44]);
  FFV1_2(w[12], w[1], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[23]);
  // Amplitude(s) for diagram number 42
  FFV1_0(w[23], w[11], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[45]);
  // Amplitude(s) for diagram number 43
  FFV1_0(w[21], w[11], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[46]);
  // Amplitude(s) for diagram number 44
  FFV1_0(w[23], w[14], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[47]);
  // Amplitude(s) for diagram number 45
  FFV1_0(w[20], w[14], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[48]);
  // Amplitude(s) for diagram number 46
  FFV1_0(w[23], w[2], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[49]);
  // Amplitude(s) for diagram number 47
  VVV1_0(w[1], w[10], w[22], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[50]);
  // Amplitude(s) for diagram number 48
  FFV1_0(w[12], w[2], w[17], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[51]);
  FFV1_0(w[12], w[2], w[19], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[52]);
  FFV1_0(w[12], w[2], w[8], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[53]);
  VVV1P0_1(w[0], w[4], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[12]);
  FFV1_2(w[3], w[12], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[22]);
  // Amplitude(s) for diagram number 49
  FFV1_0(w[22], w[9], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[54]);
  VVV1P0_1(w[12], w[5], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[23]);
  // Amplitude(s) for diagram number 50
  FFV1_0(w[3], w[9], w[23], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[55]);
  // Amplitude(s) for diagram number 51
  FFV1_0(w[13], w[9], w[12], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[56]);
  FFV1_1(w[2], w[12], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[20]);
  // Amplitude(s) for diagram number 52
  FFV1_0(w[16], w[20], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[57]);
  // Amplitude(s) for diagram number 53
  FFV1_0(w[16], w[2], w[23], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[58]);
  // Amplitude(s) for diagram number 54
  FFV1_0(w[16], w[14], w[12], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[59]);
  // Amplitude(s) for diagram number 55
  FFV1_0(w[3], w[20], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[60]);
  // Amplitude(s) for diagram number 56
  FFV1_0(w[22], w[2], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[61]);
  // Amplitude(s) for diagram number 57
  VVV1_0(w[12], w[18], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[62]);
  // Amplitude(s) for diagram number 58
  VVVV1_0(w[12], w[1], w[7], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[63]);
  VVVV3_0(w[12], w[1], w[7], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[64]);
  VVVV4_0(w[12], w[1], w[7], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[65]);
  VVV1P0_1(w[12], w[1], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[21]);
  // Amplitude(s) for diagram number 59
  VVV1_0(w[7], w[5], w[21], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[66]);
  // Amplitude(s) for diagram number 60
  VVV1_0(w[1], w[7], w[23], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[67]);
  // Amplitude(s) for diagram number 61
  FFV1_0(w[3], w[14], w[21], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[68]);
  // Amplitude(s) for diagram number 62
  FFV1_0(w[22], w[14], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[69]);
  // Amplitude(s) for diagram number 63
  FFV1_0(w[13], w[2], w[21], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[70]);
  // Amplitude(s) for diagram number 64
  FFV1_0(w[13], w[20], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[71]);
  VVV1P0_1(w[0], w[5], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[20]);
  FFV1_2(w[3], w[20], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[21]);
  // Amplitude(s) for diagram number 65
  FFV1_0(w[21], w[9], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[72]);
  VVV1P0_1(w[20], w[4], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[22]);
  // Amplitude(s) for diagram number 66
  FFV1_0(w[3], w[9], w[22], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[73]);
  // Amplitude(s) for diagram number 67
  FFV1_0(w[15], w[9], w[20], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[74]);
  FFV1_1(w[2], w[20], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[23]);
  // Amplitude(s) for diagram number 68
  FFV1_0(w[16], w[23], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[75]);
  // Amplitude(s) for diagram number 69
  FFV1_0(w[16], w[2], w[22], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[76]);
  // Amplitude(s) for diagram number 70
  FFV1_0(w[16], w[11], w[20], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[77]);
  // Amplitude(s) for diagram number 71
  FFV1_0(w[3], w[23], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[78]);
  // Amplitude(s) for diagram number 72
  FFV1_0(w[21], w[2], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[79]);
  // Amplitude(s) for diagram number 73
  VVV1_0(w[20], w[6], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[80]);
  // Amplitude(s) for diagram number 74
  VVVV1_0(w[20], w[1], w[7], w[4], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[81]);
  VVVV3_0(w[20], w[1], w[7], w[4], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[82]);
  VVVV4_0(w[20], w[1], w[7], w[4], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[83]);
  VVV1P0_1(w[20], w[1], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[12]);
  // Amplitude(s) for diagram number 75
  VVV1_0(w[7], w[4], w[12], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[84]);
  // Amplitude(s) for diagram number 76
  VVV1_0(w[1], w[7], w[22], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[85]);
  // Amplitude(s) for diagram number 77
  FFV1_0(w[3], w[11], w[12], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[86]);
  // Amplitude(s) for diagram number 78
  FFV1_0(w[21], w[11], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[87]);
  // Amplitude(s) for diagram number 79
  FFV1_0(w[15], w[2], w[12], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[88]);
  // Amplitude(s) for diagram number 80
  FFV1_0(w[15], w[23], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[89]);
  FFV1_1(w[9], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[23]);
  // Amplitude(s) for diagram number 81
  FFV1_0(w[15], w[23], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[90]);
  FFV1_2(w[15], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[12]);
  // Amplitude(s) for diagram number 82
  FFV1_0(w[12], w[9], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[91]);
  // Amplitude(s) for diagram number 83
  FFV1_0(w[13], w[23], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[92]);
  FFV1_2(w[13], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[21]);
  // Amplitude(s) for diagram number 84
  FFV1_0(w[21], w[9], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[93]);
  // Amplitude(s) for diagram number 85
  FFV1_0(w[3], w[23], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[94]);
  VVV1P0_1(w[0], w[10], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[23]);
  // Amplitude(s) for diagram number 86
  FFV1_0(w[3], w[9], w[23], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[95]);
  FFV1_2(w[16], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[22]);
  // Amplitude(s) for diagram number 87
  FFV1_0(w[22], w[11], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[96]);
  FFV1_1(w[11], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[20]);
  // Amplitude(s) for diagram number 88
  FFV1_0(w[16], w[20], w[5], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[97]);
  // Amplitude(s) for diagram number 89
  FFV1_0(w[22], w[14], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[98]);
  FFV1_1(w[14], w[0], thrust::complex<double> (cIPC[2], cIPC[3]), cIPD[0],
      cIPD[1], w[24]);
  // Amplitude(s) for diagram number 90
  FFV1_0(w[16], w[24], w[4], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[99]);
  // Amplitude(s) for diagram number 91
  FFV1_0(w[22], w[2], w[10], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[100]);
  // Amplitude(s) for diagram number 92
  FFV1_0(w[16], w[2], w[23], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[101]);
  // Amplitude(s) for diagram number 93
  VVVV1_0(w[0], w[6], w[7], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[102]);
  VVVV3_0(w[0], w[6], w[7], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[103]);
  VVVV4_0(w[0], w[6], w[7], w[5], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[104]);
  VVV1P0_1(w[0], w[6], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[22]);
  // Amplitude(s) for diagram number 94
  VVV1_0(w[7], w[5], w[22], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[105]);
  VVV1P0_1(w[0], w[7], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[25]);
  // Amplitude(s) for diagram number 95
  VVV1_0(w[6], w[5], w[25], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[106]);
  // Amplitude(s) for diagram number 96
  FFV1_0(w[3], w[14], w[22], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[107]);
  // Amplitude(s) for diagram number 97
  FFV1_0(w[3], w[24], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[108]);
  // Amplitude(s) for diagram number 98
  FFV1_0(w[13], w[2], w[22], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[109]);
  // Amplitude(s) for diagram number 99
  FFV1_0(w[21], w[2], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[110]);
  // Amplitude(s) for diagram number 100
  VVVV1_0(w[0], w[18], w[7], w[4], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[111]);
  VVVV3_0(w[0], w[18], w[7], w[4], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[112]);
  VVVV4_0(w[0], w[18], w[7], w[4], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[113]);
  VVV1P0_1(w[0], w[18], thrust::complex<double> (cIPC[0], cIPC[1]), 0., 0.,
      w[6]);
  // Amplitude(s) for diagram number 101
  VVV1_0(w[7], w[4], w[6], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[114]);
  // Amplitude(s) for diagram number 102
  VVV1_0(w[18], w[4], w[25], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[115]);
  // Amplitude(s) for diagram number 103
  FFV1_0(w[3], w[11], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[116]);
  // Amplitude(s) for diagram number 104
  FFV1_0(w[3], w[20], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[117]);
  // Amplitude(s) for diagram number 105
  FFV1_0(w[15], w[2], w[6], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[118]);
  // Amplitude(s) for diagram number 106
  FFV1_0(w[12], w[2], w[18], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[119]);
  // Amplitude(s) for diagram number 107
  VVVV1_0(w[0], w[1], w[7], w[10], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[120]);
  VVVV3_0(w[0], w[1], w[7], w[10], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[121]);
  VVVV4_0(w[0], w[1], w[7], w[10], thrust::complex<double> (cIPC[4], cIPC[5]),
      &amp[122]);
  // Amplitude(s) for diagram number 108
  VVV1_0(w[1], w[10], w[25], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[123]);
  // Amplitude(s) for diagram number 109
  VVV1_0(w[1], w[7], w[23], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[124]);
  // Amplitude(s) for diagram number 110
  FFV1_0(w[13], w[20], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[125]);
  // Amplitude(s) for diagram number 111
  FFV1_0(w[21], w[11], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[126]);
  // Amplitude(s) for diagram number 112
  FFV1_0(w[15], w[24], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[127]);
  // Amplitude(s) for diagram number 113
  FFV1_0(w[12], w[14], w[1], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[128]);
  VVVV1P0_1(w[0], w[1], w[4], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[12]);
  VVVV3P0_1(w[0], w[1], w[4], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[24]);
  VVVV4P0_1(w[0], w[1], w[4], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[21]);
  // Amplitude(s) for diagram number 114
  VVV1_0(w[12], w[7], w[5], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[129]);
  VVV1_0(w[24], w[7], w[5], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[130]);
  VVV1_0(w[21], w[7], w[5], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[131]);
  // Amplitude(s) for diagram number 115
  FFV1_0(w[3], w[14], w[12], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[132]);
  FFV1_0(w[3], w[14], w[24], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[133]);
  FFV1_0(w[3], w[14], w[21], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[134]);
  // Amplitude(s) for diagram number 116
  FFV1_0(w[13], w[2], w[12], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[135]);
  FFV1_0(w[13], w[2], w[24], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[136]);
  FFV1_0(w[13], w[2], w[21], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[137]);
  VVVV1P0_1(w[0], w[1], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[21]);
  VVVV3P0_1(w[0], w[1], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[13]);
  VVVV4P0_1(w[0], w[1], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[24]);
  // Amplitude(s) for diagram number 117
  VVV1_0(w[21], w[7], w[4], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[138]);
  VVV1_0(w[13], w[7], w[4], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[139]);
  VVV1_0(w[24], w[7], w[4], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[140]);
  // Amplitude(s) for diagram number 118
  FFV1_0(w[3], w[11], w[21], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[141]);
  FFV1_0(w[3], w[11], w[13], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[142]);
  FFV1_0(w[3], w[11], w[24], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[143]);
  // Amplitude(s) for diagram number 119
  FFV1_0(w[15], w[2], w[21], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[144]);
  FFV1_0(w[15], w[2], w[13], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[145]);
  FFV1_0(w[15], w[2], w[24], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[146]);
  VVVV1P0_1(w[0], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[24]);
  VVVV3P0_1(w[0], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[15]);
  VVVV4P0_1(w[0], w[4], w[5], thrust::complex<double> (cIPC[4], cIPC[5]), 0.,
      0., w[13]);
  // Amplitude(s) for diagram number 120
  FFV1_0(w[3], w[9], w[24], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[147]);
  FFV1_0(w[3], w[9], w[15], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[148]);
  FFV1_0(w[3], w[9], w[13], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[149]);
  // Amplitude(s) for diagram number 121
  FFV1_0(w[16], w[2], w[24], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[150]);
  FFV1_0(w[16], w[2], w[15], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[151]);
  FFV1_0(w[16], w[2], w[13], thrust::complex<double> (cIPC[2], cIPC[3]),
      &amp[152]);
  // Amplitude(s) for diagram number 122
  VVV1_0(w[24], w[1], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[153]);
  VVV1_0(w[15], w[1], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[154]);
  VVV1_0(w[13], w[1], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[155]);
  // Amplitude(s) for diagram number 123
  VVV1_0(w[0], w[17], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[156]);
  VVV1_0(w[0], w[19], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[157]);
  VVV1_0(w[0], w[8], w[7], thrust::complex<double> (cIPC[0], cIPC[1]),
      &amp[158]);
  // double CPPProcess::matrix_1_gg_ttxgg() {
  int i, j; 
  // Local variables

  // const int ngraphs = 2;
  const int ncolor = 24; 
  thrust::complex<double> ztemp; 
  thrust::complex<double> jamp[ncolor]; 
  // The color matrix;
  static const double denom[ncolor] = {54, 54, 54, 54, 54, 54, 54, 54, 54, 54,
      54, 54, 54, 54, 54, 54, 54, 54, 54, 54, 54, 54, 54, 54};
  static const double cf[ncolor][ncolor] = {{512, -64, -64, 8, 8, 80, -64, 8,
      8, -1, -1, -10, 8, -1, 80, -10, 71, 62, -1, -10, -10, 62, 62, -28}, {-64,
      512, 8, 80, -64, 8, 8, -64, -1, -10, 8, -1, -1, -10, -10, 62, 62, -28, 8,
      -1, 80, -10, 71, 62}, {-64, 8, 512, -64, 80, 8, 8, -1, 80, -10, 71, 62,
      -64, 8, 8, -1, -1, -10, -10, -1, 62, -28, -10, 62}, {8, 80, -64, 512, 8,
      -64, -1, -10, -10, 62, 62, -28, 8, -64, -1, -10, 8, -1, -1, 8, 71, 62,
      80, -10}, {8, -64, 80, 8, 512, -64, -1, 8, 71, 62, 80, -10, -10, -1, 62,
      -28, -10, 62, -64, 8, 8, -1, -1, -10}, {80, 8, 8, -64, -64, 512, -10, -1,
      62, -28, -10, 62, -1, 8, 71, 62, 80, -10, 8, -64, -1, -10, 8, -1}, {-64,
      8, 8, -1, -1, -10, 512, -64, -64, 8, 8, 80, 80, -10, 8, -1, 62, 71, -10,
      62, -1, -10, -28, 62}, {8, -64, -1, -10, 8, -1, -64, 512, 8, 80, -64, 8,
      -10, 62, -1, -10, -28, 62, 80, -10, 8, -1, 62, 71}, {8, -1, 80, -10, 71,
      62, -64, 8, 512, -64, 80, 8, 8, -1, -64, 8, -10, -1, 62, -28, -10, -1,
      62, -10}, {-1, -10, -10, 62, 62, -28, 8, 80, -64, 512, 8, -64, -1, -10,
      8, -64, -1, 8, 71, 62, -1, 8, -10, 80}, {-1, 8, 71, 62, 80, -10, 8, -64,
      80, 8, 512, -64, 62, -28, -10, -1, 62, -10, 8, -1, -64, 8, -10, -1},
      {-10, -1, 62, -28, -10, 62, 80, 8, 8, -64, -64, 512, 71, 62, -1, 8, -10,
      80, -1, -10, 8, -64, -1, 8}, {8, -1, -64, 8, -10, -1, 80, -10, 8, -1, 62,
      71, 512, -64, -64, 8, 8, 80, 62, -10, -28, 62, -1, -10}, {-1, -10, 8,
      -64, -1, 8, -10, 62, -1, -10, -28, 62, -64, 512, 8, 80, -64, 8, -10, 80,
      62, 71, 8, -1}, {80, -10, 8, -1, 62, 71, 8, -1, -64, 8, -10, -1, -64, 8,
      512, -64, 80, 8, -28, 62, 62, -10, -10, -1}, {-10, 62, -1, -10, -28, 62,
      -1, -10, 8, -64, -1, 8, 8, 80, -64, 512, 8, -64, 62, 71, -10, 80, -1, 8},
      {71, 62, -1, 8, -10, 80, 62, -28, -10, -1, 62, -10, 8, -64, 80, 8, 512,
      -64, -1, 8, -10, -1, -64, 8}, {62, -28, -10, -1, 62, -10, 71, 62, -1, 8,
      -10, 80, 80, 8, 8, -64, -64, 512, -10, -1, -1, 8, 8, -64}, {-1, 8, -10,
      -1, -64, 8, -10, 80, 62, 71, 8, -1, 62, -10, -28, 62, -1, -10, 512, -64,
      -64, 8, 8, 80}, {-10, -1, -1, 8, 8, -64, 62, -10, -28, 62, -1, -10, -10,
      80, 62, 71, 8, -1, -64, 512, 8, 80, -64, 8}, {-10, 80, 62, 71, 8, -1, -1,
      8, -10, -1, -64, 8, -28, 62, 62, -10, -10, -1, -64, 8, 512, -64, 80, 8},
      {62, -10, -28, 62, -1, -10, -10, -1, -1, 8, 8, -64, 62, 71, -10, 80, -1,
      8, 8, 80, -64, 512, 8, -64}, {62, 71, -10, 80, -1, 8, -28, 62, 62, -10,
      -10, -1, -1, 8, -10, -1, -64, 8, 8, -64, 80, 8, 512, -64}, {-28, 62, 62,
      -10, -10, -1, 62, 71, -10, 80, -1, 8, -10, -1, -1, 8, 8, -64, 80, 8, 8,
      -64, -64, 512}};

  // Calculate color flows
  jamp[0] = +thrust::complex<double> (0, 1) * amp[0] + thrust::complex<double>
      (0, 1) * amp[1] + thrust::complex<double> (0, 1) * amp[3] +
      thrust::complex<double> (0, 1) * amp[5] + thrust::complex<double> (0, 1)
      * amp[14] + amp[15] + amp[16] + amp[21] + thrust::complex<double> (0, 1)
      * amp[23] - amp[29] + thrust::complex<double> (0, 1) * amp[31] + amp[32]
      + amp[33] - amp[35] + thrust::complex<double> (0, 1) * amp[102] +
      thrust::complex<double> (0, 1) * amp[103] + thrust::complex<double> (0,
      1) * amp[105] + thrust::complex<double> (0, 1) * amp[106] + amp[109] +
      thrust::complex<double> (0, 1) * amp[120] + thrust::complex<double> (0,
      1) * amp[121] + thrust::complex<double> (0, 1) * amp[123] +
      thrust::complex<double> (0, 1) * amp[129] - thrust::complex<double> (0,
      1) * amp[131] + amp[135] - amp[137] - thrust::complex<double> (0, 1) *
      amp[156] + thrust::complex<double> (0, 1) * amp[158];
  jamp[1] = -thrust::complex<double> (0, 1) * amp[0] + thrust::complex<double>
      (0, 1) * amp[2] + thrust::complex<double> (0, 1) * amp[4] -
      thrust::complex<double> (0, 1) * amp[5] + thrust::complex<double> (0, 1)
      * amp[12] + amp[13] - amp[16] + amp[24] + thrust::complex<double> (0, 1)
      * amp[26] - amp[27] - thrust::complex<double> (0, 1) * amp[31] - amp[32]
      - amp[33] - amp[34] + thrust::complex<double> (0, 1) * amp[111] +
      thrust::complex<double> (0, 1) * amp[112] + thrust::complex<double> (0,
      1) * amp[114] + thrust::complex<double> (0, 1) * amp[115] + amp[118] -
      thrust::complex<double> (0, 1) * amp[120] - thrust::complex<double> (0,
      1) * amp[121] - thrust::complex<double> (0, 1) * amp[123] +
      thrust::complex<double> (0, 1) * amp[138] - thrust::complex<double> (0,
      1) * amp[140] + amp[144] - amp[146] + thrust::complex<double> (0, 1) *
      amp[156] + thrust::complex<double> (0, 1) * amp[157];
  jamp[2] = -amp[21] - thrust::complex<double> (0, 1) * amp[23] - amp[24] +
      thrust::complex<double> (0, 1) * amp[25] - amp[30] + amp[34] + amp[35] +
      amp[60] - thrust::complex<double> (0, 1) * amp[62] +
      thrust::complex<double> (0, 1) * amp[63] + thrust::complex<double> (0, 1)
      * amp[64] + thrust::complex<double> (0, 1) * amp[66] + amp[70] +
      thrust::complex<double> (0, 1) * amp[71] - thrust::complex<double> (0, 1)
      * amp[102] - thrust::complex<double> (0, 1) * amp[103] -
      thrust::complex<double> (0, 1) * amp[105] - thrust::complex<double> (0,
      1) * amp[106] - amp[109] - thrust::complex<double> (0, 1) * amp[112] -
      thrust::complex<double> (0, 1) * amp[113] - thrust::complex<double> (0,
      1) * amp[115] - thrust::complex<double> (0, 1) * amp[129] -
      thrust::complex<double> (0, 1) * amp[130] - amp[135] - amp[136] -
      thrust::complex<double> (0, 1) * amp[157] - thrust::complex<double> (0,
      1) * amp[158];
  jamp[3] = -amp[18] + thrust::complex<double> (0, 1) * amp[20] + amp[24] -
      thrust::complex<double> (0, 1) * amp[25] - amp[32] - amp[33] - amp[34] +
      thrust::complex<double> (0, 1) * amp[57] + amp[58] - amp[60] +
      thrust::complex<double> (0, 1) * amp[62] - thrust::complex<double> (0, 1)
      * amp[64] - thrust::complex<double> (0, 1) * amp[65] -
      thrust::complex<double> (0, 1) * amp[67] + amp[101] +
      thrust::complex<double> (0, 1) * amp[112] + thrust::complex<double> (0,
      1) * amp[113] + thrust::complex<double> (0, 1) * amp[115] -
      thrust::complex<double> (0, 1) * amp[121] - thrust::complex<double> (0,
      1) * amp[122] - thrust::complex<double> (0, 1) * amp[123] -
      thrust::complex<double> (0, 1) * amp[124] + amp[150] - amp[152] -
      thrust::complex<double> (0, 1) * amp[153] + thrust::complex<double> (0,
      1) * amp[155] + thrust::complex<double> (0, 1) * amp[156] +
      thrust::complex<double> (0, 1) * amp[157];
  jamp[4] = -amp[21] + thrust::complex<double> (0, 1) * amp[22] - amp[24] -
      thrust::complex<double> (0, 1) * amp[26] - amp[28] + amp[34] + amp[35] +
      amp[78] - thrust::complex<double> (0, 1) * amp[80] +
      thrust::complex<double> (0, 1) * amp[81] + thrust::complex<double> (0, 1)
      * amp[82] + thrust::complex<double> (0, 1) * amp[84] + amp[88] +
      thrust::complex<double> (0, 1) * amp[89] - thrust::complex<double> (0, 1)
      * amp[103] - thrust::complex<double> (0, 1) * amp[104] -
      thrust::complex<double> (0, 1) * amp[106] - thrust::complex<double> (0,
      1) * amp[111] - thrust::complex<double> (0, 1) * amp[112] -
      thrust::complex<double> (0, 1) * amp[114] - thrust::complex<double> (0,
      1) * amp[115] - amp[118] - thrust::complex<double> (0, 1) * amp[138] -
      thrust::complex<double> (0, 1) * amp[139] - amp[144] - amp[145] -
      thrust::complex<double> (0, 1) * amp[157] - thrust::complex<double> (0,
      1) * amp[158];
  jamp[5] = -amp[19] - thrust::complex<double> (0, 1) * amp[20] + amp[21] -
      thrust::complex<double> (0, 1) * amp[22] + amp[32] + amp[33] - amp[35] +
      thrust::complex<double> (0, 1) * amp[75] + amp[76] - amp[78] +
      thrust::complex<double> (0, 1) * amp[80] - thrust::complex<double> (0, 1)
      * amp[82] - thrust::complex<double> (0, 1) * amp[83] -
      thrust::complex<double> (0, 1) * amp[85] - amp[101] +
      thrust::complex<double> (0, 1) * amp[103] + thrust::complex<double> (0,
      1) * amp[104] + thrust::complex<double> (0, 1) * amp[106] +
      thrust::complex<double> (0, 1) * amp[121] + thrust::complex<double> (0,
      1) * amp[122] + thrust::complex<double> (0, 1) * amp[123] +
      thrust::complex<double> (0, 1) * amp[124] - amp[150] - amp[151] +
      thrust::complex<double> (0, 1) * amp[153] + thrust::complex<double> (0,
      1) * amp[154] - thrust::complex<double> (0, 1) * amp[156] +
      thrust::complex<double> (0, 1) * amp[158];
  jamp[6] = -thrust::complex<double> (0, 1) * amp[0] - thrust::complex<double>
      (0, 1) * amp[1] - thrust::complex<double> (0, 1) * amp[3] -
      thrust::complex<double> (0, 1) * amp[5] - thrust::complex<double> (0, 1)
      * amp[14] - amp[15] - amp[16] + amp[55] + thrust::complex<double> (0, 1)
      * amp[56] - thrust::complex<double> (0, 1) * amp[63] +
      thrust::complex<double> (0, 1) * amp[65] - thrust::complex<double> (0, 1)
      * amp[66] + thrust::complex<double> (0, 1) * amp[67] - amp[70] - amp[92]
      + thrust::complex<double> (0, 1) * amp[94] + amp[95] -
      thrust::complex<double> (0, 1) * amp[120] + thrust::complex<double> (0,
      1) * amp[122] + thrust::complex<double> (0, 1) * amp[124] +
      thrust::complex<double> (0, 1) * amp[130] + thrust::complex<double> (0,
      1) * amp[131] + amp[136] + amp[137] + amp[147] - amp[149] +
      thrust::complex<double> (0, 1) * amp[153] - thrust::complex<double> (0,
      1) * amp[155];
  jamp[7] = +thrust::complex<double> (0, 1) * amp[0] - thrust::complex<double>
      (0, 1) * amp[2] - thrust::complex<double> (0, 1) * amp[4] +
      thrust::complex<double> (0, 1) * amp[5] - thrust::complex<double> (0, 1)
      * amp[12] - amp[13] + amp[16] + amp[73] + thrust::complex<double> (0, 1)
      * amp[74] - thrust::complex<double> (0, 1) * amp[81] +
      thrust::complex<double> (0, 1) * amp[83] - thrust::complex<double> (0, 1)
      * amp[84] + thrust::complex<double> (0, 1) * amp[85] - amp[88] - amp[90]
      - thrust::complex<double> (0, 1) * amp[94] - amp[95] +
      thrust::complex<double> (0, 1) * amp[120] - thrust::complex<double> (0,
      1) * amp[122] - thrust::complex<double> (0, 1) * amp[124] +
      thrust::complex<double> (0, 1) * amp[139] + thrust::complex<double> (0,
      1) * amp[140] + amp[145] + amp[146] - amp[147] - amp[148] -
      thrust::complex<double> (0, 1) * amp[153] - thrust::complex<double> (0,
      1) * amp[154];
  jamp[8] = -amp[55] - thrust::complex<double> (0, 1) * amp[56] +
      thrust::complex<double> (0, 1) * amp[63] - thrust::complex<double> (0, 1)
      * amp[65] + thrust::complex<double> (0, 1) * amp[66] -
      thrust::complex<double> (0, 1) * amp[67] + amp[70] +
      thrust::complex<double> (0, 1) * amp[72] - amp[73] + amp[79] +
      thrust::complex<double> (0, 1) * amp[80] - thrust::complex<double> (0, 1)
      * amp[82] - thrust::complex<double> (0, 1) * amp[83] -
      thrust::complex<double> (0, 1) * amp[85] - amp[93] -
      thrust::complex<double> (0, 1) * amp[102] + thrust::complex<double> (0,
      1) * amp[104] - thrust::complex<double> (0, 1) * amp[105] - amp[109] +
      thrust::complex<double> (0, 1) * amp[110] - thrust::complex<double> (0,
      1) * amp[129] - thrust::complex<double> (0, 1) * amp[130] - amp[135] -
      amp[136] + amp[148] + amp[149] + thrust::complex<double> (0, 1) *
      amp[154] + thrust::complex<double> (0, 1) * amp[155];
  jamp[9] = -amp[37] + thrust::complex<double> (0, 1) * amp[38] + amp[39] +
      thrust::complex<double> (0, 1) * amp[40] + amp[50] + amp[51] - amp[53] -
      thrust::complex<double> (0, 1) * amp[72] + amp[73] - amp[79] -
      thrust::complex<double> (0, 1) * amp[80] + thrust::complex<double> (0, 1)
      * amp[82] + thrust::complex<double> (0, 1) * amp[83] +
      thrust::complex<double> (0, 1) * amp[85] - amp[95] -
      thrust::complex<double> (0, 1) * amp[103] - thrust::complex<double> (0,
      1) * amp[104] - thrust::complex<double> (0, 1) * amp[106] -
      thrust::complex<double> (0, 1) * amp[121] - thrust::complex<double> (0,
      1) * amp[122] - thrust::complex<double> (0, 1) * amp[123] -
      thrust::complex<double> (0, 1) * amp[124] - amp[147] - amp[148] -
      thrust::complex<double> (0, 1) * amp[153] - thrust::complex<double> (0,
      1) * amp[154] + thrust::complex<double> (0, 1) * amp[156] -
      thrust::complex<double> (0, 1) * amp[158];
  jamp[10] = +thrust::complex<double> (0, 1) * amp[54] - amp[55] + amp[61] +
      thrust::complex<double> (0, 1) * amp[62] - thrust::complex<double> (0, 1)
      * amp[64] - thrust::complex<double> (0, 1) * amp[65] -
      thrust::complex<double> (0, 1) * amp[67] - amp[73] -
      thrust::complex<double> (0, 1) * amp[74] + thrust::complex<double> (0, 1)
      * amp[81] - thrust::complex<double> (0, 1) * amp[83] +
      thrust::complex<double> (0, 1) * amp[84] - thrust::complex<double> (0, 1)
      * amp[85] + amp[88] - amp[91] - thrust::complex<double> (0, 1) * amp[111]
      + thrust::complex<double> (0, 1) * amp[113] - thrust::complex<double> (0,
      1) * amp[114] - amp[118] + thrust::complex<double> (0, 1) * amp[119] -
      thrust::complex<double> (0, 1) * amp[138] - thrust::complex<double> (0,
      1) * amp[139] - amp[144] - amp[145] + amp[148] + amp[149] +
      thrust::complex<double> (0, 1) * amp[154] + thrust::complex<double> (0,
      1) * amp[155];
  jamp[11] = -amp[36] - thrust::complex<double> (0, 1) * amp[38] + amp[42] +
      thrust::complex<double> (0, 1) * amp[43] - amp[50] - amp[51] - amp[52] -
      thrust::complex<double> (0, 1) * amp[54] + amp[55] - amp[61] -
      thrust::complex<double> (0, 1) * amp[62] + thrust::complex<double> (0, 1)
      * amp[64] + thrust::complex<double> (0, 1) * amp[65] +
      thrust::complex<double> (0, 1) * amp[67] + amp[95] -
      thrust::complex<double> (0, 1) * amp[112] - thrust::complex<double> (0,
      1) * amp[113] - thrust::complex<double> (0, 1) * amp[115] +
      thrust::complex<double> (0, 1) * amp[121] + thrust::complex<double> (0,
      1) * amp[122] + thrust::complex<double> (0, 1) * amp[123] +
      thrust::complex<double> (0, 1) * amp[124] + amp[147] - amp[149] +
      thrust::complex<double> (0, 1) * amp[153] - thrust::complex<double> (0,
      1) * amp[155] - thrust::complex<double> (0, 1) * amp[156] -
      thrust::complex<double> (0, 1) * amp[157];
  jamp[12] = -thrust::complex<double> (0, 1) * amp[1] - thrust::complex<double>
      (0, 1) * amp[2] - thrust::complex<double> (0, 1) * amp[3] -
      thrust::complex<double> (0, 1) * amp[4] + amp[7] +
      thrust::complex<double> (0, 1) * amp[8] - amp[15] - amp[60] +
      thrust::complex<double> (0, 1) * amp[62] - thrust::complex<double> (0, 1)
      * amp[63] - thrust::complex<double> (0, 1) * amp[64] -
      thrust::complex<double> (0, 1) * amp[66] - amp[70] -
      thrust::complex<double> (0, 1) * amp[71] - thrust::complex<double> (0, 1)
      * amp[111] + thrust::complex<double> (0, 1) * amp[113] -
      thrust::complex<double> (0, 1) * amp[114] + amp[116] +
      thrust::complex<double> (0, 1) * amp[117] - amp[125] +
      thrust::complex<double> (0, 1) * amp[130] + thrust::complex<double> (0,
      1) * amp[131] + amp[136] + amp[137] - thrust::complex<double> (0, 1) *
      amp[138] + thrust::complex<double> (0, 1) * amp[140] + amp[141] -
      amp[143];
  jamp[13] = -thrust::complex<double> (0, 1) * amp[57] - amp[58] + amp[60] -
      thrust::complex<double> (0, 1) * amp[62] + thrust::complex<double> (0, 1)
      * amp[64] + thrust::complex<double> (0, 1) * amp[65] +
      thrust::complex<double> (0, 1) * amp[67] - amp[76] +
      thrust::complex<double> (0, 1) * amp[77] - thrust::complex<double> (0, 1)
      * amp[81] + thrust::complex<double> (0, 1) * amp[83] -
      thrust::complex<double> (0, 1) * amp[84] + thrust::complex<double> (0, 1)
      * amp[85] + amp[86] - amp[97] + thrust::complex<double> (0, 1) * amp[111]
      - thrust::complex<double> (0, 1) * amp[113] + thrust::complex<double> (0,
      1) * amp[114] - amp[116] - thrust::complex<double> (0, 1) * amp[117] +
      thrust::complex<double> (0, 1) * amp[138] + thrust::complex<double> (0,
      1) * amp[139] - amp[141] - amp[142] + amp[151] + amp[152] -
      thrust::complex<double> (0, 1) * amp[154] - thrust::complex<double> (0,
      1) * amp[155];
  jamp[14] = +thrust::complex<double> (0, 1) * amp[1] + thrust::complex<double>
      (0, 1) * amp[2] + thrust::complex<double> (0, 1) * amp[3] +
      thrust::complex<double> (0, 1) * amp[4] - amp[7] -
      thrust::complex<double> (0, 1) * amp[8] + amp[15] - amp[79] -
      thrust::complex<double> (0, 1) * amp[80] + thrust::complex<double> (0, 1)
      * amp[81] + thrust::complex<double> (0, 1) * amp[82] +
      thrust::complex<double> (0, 1) * amp[84] - amp[86] +
      thrust::complex<double> (0, 1) * amp[87] + thrust::complex<double> (0, 1)
      * amp[102] - thrust::complex<double> (0, 1) * amp[104] +
      thrust::complex<double> (0, 1) * amp[105] + amp[109] -
      thrust::complex<double> (0, 1) * amp[110] - amp[126] +
      thrust::complex<double> (0, 1) * amp[129] - thrust::complex<double> (0,
      1) * amp[131] + amp[135] - amp[137] - thrust::complex<double> (0, 1) *
      amp[139] - thrust::complex<double> (0, 1) * amp[140] + amp[142] +
      amp[143];
  jamp[15] = -amp[39] - thrust::complex<double> (0, 1) * amp[40] - amp[42] +
      thrust::complex<double> (0, 1) * amp[44] - amp[46] + amp[52] + amp[53] +
      amp[79] + thrust::complex<double> (0, 1) * amp[80] -
      thrust::complex<double> (0, 1) * amp[81] - thrust::complex<double> (0, 1)
      * amp[82] - thrust::complex<double> (0, 1) * amp[84] + amp[86] -
      thrust::complex<double> (0, 1) * amp[87] + thrust::complex<double> (0, 1)
      * amp[103] + thrust::complex<double> (0, 1) * amp[104] +
      thrust::complex<double> (0, 1) * amp[106] + thrust::complex<double> (0,
      1) * amp[111] + thrust::complex<double> (0, 1) * amp[112] +
      thrust::complex<double> (0, 1) * amp[114] + thrust::complex<double> (0,
      1) * amp[115] - amp[116] + thrust::complex<double> (0, 1) * amp[138] +
      thrust::complex<double> (0, 1) * amp[139] - amp[141] - amp[142] +
      thrust::complex<double> (0, 1) * amp[157] + thrust::complex<double> (0,
      1) * amp[158];
  jamp[16] = -thrust::complex<double> (0, 1) * amp[0] + thrust::complex<double>
      (0, 1) * amp[2] + thrust::complex<double> (0, 1) * amp[4] -
      thrust::complex<double> (0, 1) * amp[5] + thrust::complex<double> (0, 1)
      * amp[6] - amp[7] + amp[17] + amp[76] - thrust::complex<double> (0, 1) *
      amp[77] + thrust::complex<double> (0, 1) * amp[81] -
      thrust::complex<double> (0, 1) * amp[83] + thrust::complex<double> (0, 1)
      * amp[84] - thrust::complex<double> (0, 1) * amp[85] - amp[86] - amp[96]
      + thrust::complex<double> (0, 1) * amp[100] - amp[101] -
      thrust::complex<double> (0, 1) * amp[120] + thrust::complex<double> (0,
      1) * amp[122] + thrust::complex<double> (0, 1) * amp[124] -
      thrust::complex<double> (0, 1) * amp[139] - thrust::complex<double> (0,
      1) * amp[140] + amp[142] + amp[143] - amp[150] - amp[151] +
      thrust::complex<double> (0, 1) * amp[153] + thrust::complex<double> (0,
      1) * amp[154];
  jamp[17] = +thrust::complex<double> (0, 1) * amp[0] - thrust::complex<double>
      (0, 1) * amp[2] - thrust::complex<double> (0, 1) * amp[4] +
      thrust::complex<double> (0, 1) * amp[5] - thrust::complex<double> (0, 1)
      * amp[6] + amp[7] - amp[17] + amp[42] - thrust::complex<double> (0, 1) *
      amp[44] - amp[45] + thrust::complex<double> (0, 1) * amp[49] - amp[50] -
      amp[51] - amp[52] - thrust::complex<double> (0, 1) * amp[111] -
      thrust::complex<double> (0, 1) * amp[112] - thrust::complex<double> (0,
      1) * amp[114] - thrust::complex<double> (0, 1) * amp[115] + amp[116] +
      thrust::complex<double> (0, 1) * amp[120] + thrust::complex<double> (0,
      1) * amp[121] + thrust::complex<double> (0, 1) * amp[123] -
      thrust::complex<double> (0, 1) * amp[138] + thrust::complex<double> (0,
      1) * amp[140] + amp[141] - amp[143] - thrust::complex<double> (0, 1) *
      amp[156] - thrust::complex<double> (0, 1) * amp[157];
  jamp[18] = -thrust::complex<double> (0, 1) * amp[1] - thrust::complex<double>
      (0, 1) * amp[2] - thrust::complex<double> (0, 1) * amp[3] -
      thrust::complex<double> (0, 1) * amp[4] + amp[10] +
      thrust::complex<double> (0, 1) * amp[11] - amp[13] - amp[78] +
      thrust::complex<double> (0, 1) * amp[80] - thrust::complex<double> (0, 1)
      * amp[81] - thrust::complex<double> (0, 1) * amp[82] -
      thrust::complex<double> (0, 1) * amp[84] - amp[88] -
      thrust::complex<double> (0, 1) * amp[89] - thrust::complex<double> (0, 1)
      * amp[102] + thrust::complex<double> (0, 1) * amp[104] -
      thrust::complex<double> (0, 1) * amp[105] + amp[107] +
      thrust::complex<double> (0, 1) * amp[108] - amp[127] -
      thrust::complex<double> (0, 1) * amp[129] + thrust::complex<double> (0,
      1) * amp[131] + amp[132] - amp[134] + thrust::complex<double> (0, 1) *
      amp[139] + thrust::complex<double> (0, 1) * amp[140] + amp[145] +
      amp[146];
  jamp[19] = -amp[58] + thrust::complex<double> (0, 1) * amp[59] -
      thrust::complex<double> (0, 1) * amp[63] + thrust::complex<double> (0, 1)
      * amp[65] - thrust::complex<double> (0, 1) * amp[66] +
      thrust::complex<double> (0, 1) * amp[67] + amp[68] -
      thrust::complex<double> (0, 1) * amp[75] - amp[76] + amp[78] -
      thrust::complex<double> (0, 1) * amp[80] + thrust::complex<double> (0, 1)
      * amp[82] + thrust::complex<double> (0, 1) * amp[83] +
      thrust::complex<double> (0, 1) * amp[85] - amp[99] +
      thrust::complex<double> (0, 1) * amp[102] - thrust::complex<double> (0,
      1) * amp[104] + thrust::complex<double> (0, 1) * amp[105] - amp[107] -
      thrust::complex<double> (0, 1) * amp[108] + thrust::complex<double> (0,
      1) * amp[129] + thrust::complex<double> (0, 1) * amp[130] - amp[132] -
      amp[133] + amp[151] + amp[152] - thrust::complex<double> (0, 1) *
      amp[154] - thrust::complex<double> (0, 1) * amp[155];
  jamp[20] = +thrust::complex<double> (0, 1) * amp[1] + thrust::complex<double>
      (0, 1) * amp[2] + thrust::complex<double> (0, 1) * amp[3] +
      thrust::complex<double> (0, 1) * amp[4] - amp[10] -
      thrust::complex<double> (0, 1) * amp[11] + amp[13] - amp[61] -
      thrust::complex<double> (0, 1) * amp[62] + thrust::complex<double> (0, 1)
      * amp[63] + thrust::complex<double> (0, 1) * amp[64] +
      thrust::complex<double> (0, 1) * amp[66] - amp[68] +
      thrust::complex<double> (0, 1) * amp[69] + thrust::complex<double> (0, 1)
      * amp[111] - thrust::complex<double> (0, 1) * amp[113] +
      thrust::complex<double> (0, 1) * amp[114] + amp[118] -
      thrust::complex<double> (0, 1) * amp[119] - amp[128] -
      thrust::complex<double> (0, 1) * amp[130] - thrust::complex<double> (0,
      1) * amp[131] + amp[133] + amp[134] + thrust::complex<double> (0, 1) *
      amp[138] - thrust::complex<double> (0, 1) * amp[140] + amp[144] -
      amp[146];
  jamp[21] = -amp[39] + thrust::complex<double> (0, 1) * amp[41] - amp[42] -
      thrust::complex<double> (0, 1) * amp[43] - amp[48] + amp[52] + amp[53] +
      amp[61] + thrust::complex<double> (0, 1) * amp[62] -
      thrust::complex<double> (0, 1) * amp[63] - thrust::complex<double> (0, 1)
      * amp[64] - thrust::complex<double> (0, 1) * amp[66] + amp[68] -
      thrust::complex<double> (0, 1) * amp[69] + thrust::complex<double> (0, 1)
      * amp[102] + thrust::complex<double> (0, 1) * amp[103] +
      thrust::complex<double> (0, 1) * amp[105] + thrust::complex<double> (0,
      1) * amp[106] - amp[107] + thrust::complex<double> (0, 1) * amp[112] +
      thrust::complex<double> (0, 1) * amp[113] + thrust::complex<double> (0,
      1) * amp[115] + thrust::complex<double> (0, 1) * amp[129] +
      thrust::complex<double> (0, 1) * amp[130] - amp[132] - amp[133] +
      thrust::complex<double> (0, 1) * amp[157] + thrust::complex<double> (0,
      1) * amp[158];
  jamp[22] = +thrust::complex<double> (0, 1) * amp[0] + thrust::complex<double>
      (0, 1) * amp[1] + thrust::complex<double> (0, 1) * amp[3] +
      thrust::complex<double> (0, 1) * amp[5] + thrust::complex<double> (0, 1)
      * amp[9] - amp[10] - amp[17] + amp[58] - thrust::complex<double> (0, 1) *
      amp[59] + thrust::complex<double> (0, 1) * amp[63] -
      thrust::complex<double> (0, 1) * amp[65] + thrust::complex<double> (0, 1)
      * amp[66] - thrust::complex<double> (0, 1) * amp[67] - amp[68] - amp[98]
      - thrust::complex<double> (0, 1) * amp[100] + amp[101] +
      thrust::complex<double> (0, 1) * amp[120] - thrust::complex<double> (0,
      1) * amp[122] - thrust::complex<double> (0, 1) * amp[124] -
      thrust::complex<double> (0, 1) * amp[130] - thrust::complex<double> (0,
      1) * amp[131] + amp[133] + amp[134] + amp[150] - amp[152] -
      thrust::complex<double> (0, 1) * amp[153] + thrust::complex<double> (0,
      1) * amp[155];
  jamp[23] = -thrust::complex<double> (0, 1) * amp[0] - thrust::complex<double>
      (0, 1) * amp[1] - thrust::complex<double> (0, 1) * amp[3] -
      thrust::complex<double> (0, 1) * amp[5] - thrust::complex<double> (0, 1)
      * amp[9] + amp[10] + amp[17] + amp[39] - thrust::complex<double> (0, 1) *
      amp[41] - amp[47] - thrust::complex<double> (0, 1) * amp[49] + amp[50] +
      amp[51] - amp[53] - thrust::complex<double> (0, 1) * amp[102] -
      thrust::complex<double> (0, 1) * amp[103] - thrust::complex<double> (0,
      1) * amp[105] - thrust::complex<double> (0, 1) * amp[106] + amp[107] -
      thrust::complex<double> (0, 1) * amp[120] - thrust::complex<double> (0,
      1) * amp[121] - thrust::complex<double> (0, 1) * amp[123] -
      thrust::complex<double> (0, 1) * amp[129] + thrust::complex<double> (0,
      1) * amp[131] + amp[132] - amp[134] + thrust::complex<double> (0, 1) *
      amp[156] - thrust::complex<double> (0, 1) * amp[158];

  // Sum and square the color flows to get the matrix element
  for(i = 0; i < ncolor; i++ )
  {
    ztemp = 0.; 
    for(j = 0; j < ncolor; j++ )
      ztemp = ztemp + cf[i][j] * jamp[j]; 
    matrix = matrix + (ztemp * conj(jamp[i])).real()/denom[i]; 
  }

  // Store the leading color flows for choice of color
  // for(i=0;i < ncolor; i++)
  // jamp2[0][i] += real(jamp[i]*conj(jamp[i]));

}



CPPProcess::CPPProcess(int numiterations, int gpublocks, int gputhreads, 
bool verbose, bool debug)
: m_numiterations(numiterations), gpu_nblocks(gpublocks), 
gpu_nthreads(gputhreads), dim(gpu_nblocks * gpu_nthreads) 
{


  // Helicities for the process - nodim
  static const int tHel[ncomb][nexternal] = {{-1, -1, -1, -1, -1, -1}, {-1, -1,
      -1, -1, -1, 1}, {-1, -1, -1, -1, 1, -1}, {-1, -1, -1, -1, 1, 1}, {-1, -1,
      -1, 1, -1, -1}, {-1, -1, -1, 1, -1, 1}, {-1, -1, -1, 1, 1, -1}, {-1, -1,
      -1, 1, 1, 1}, {-1, -1, 1, -1, -1, -1}, {-1, -1, 1, -1, -1, 1}, {-1, -1,
      1, -1, 1, -1}, {-1, -1, 1, -1, 1, 1}, {-1, -1, 1, 1, -1, -1}, {-1, -1, 1,
      1, -1, 1}, {-1, -1, 1, 1, 1, -1}, {-1, -1, 1, 1, 1, 1}, {-1, 1, -1, -1,
      -1, -1}, {-1, 1, -1, -1, -1, 1}, {-1, 1, -1, -1, 1, -1}, {-1, 1, -1, -1,
      1, 1}, {-1, 1, -1, 1, -1, -1}, {-1, 1, -1, 1, -1, 1}, {-1, 1, -1, 1, 1,
      -1}, {-1, 1, -1, 1, 1, 1}, {-1, 1, 1, -1, -1, -1}, {-1, 1, 1, -1, -1, 1},
      {-1, 1, 1, -1, 1, -1}, {-1, 1, 1, -1, 1, 1}, {-1, 1, 1, 1, -1, -1}, {-1,
      1, 1, 1, -1, 1}, {-1, 1, 1, 1, 1, -1}, {-1, 1, 1, 1, 1, 1}, {1, -1, -1,
      -1, -1, -1}, {1, -1, -1, -1, -1, 1}, {1, -1, -1, -1, 1, -1}, {1, -1, -1,
      -1, 1, 1}, {1, -1, -1, 1, -1, -1}, {1, -1, -1, 1, -1, 1}, {1, -1, -1, 1,
      1, -1}, {1, -1, -1, 1, 1, 1}, {1, -1, 1, -1, -1, -1}, {1, -1, 1, -1, -1,
      1}, {1, -1, 1, -1, 1, -1}, {1, -1, 1, -1, 1, 1}, {1, -1, 1, 1, -1, -1},
      {1, -1, 1, 1, -1, 1}, {1, -1, 1, 1, 1, -1}, {1, -1, 1, 1, 1, 1}, {1, 1,
      -1, -1, -1, -1}, {1, 1, -1, -1, -1, 1}, {1, 1, -1, -1, 1, -1}, {1, 1, -1,
      -1, 1, 1}, {1, 1, -1, 1, -1, -1}, {1, 1, -1, 1, -1, 1}, {1, 1, -1, 1, 1,
      -1}, {1, 1, -1, 1, 1, 1}, {1, 1, 1, -1, -1, -1}, {1, 1, 1, -1, -1, 1},
      {1, 1, 1, -1, 1, -1}, {1, 1, 1, -1, 1, 1}, {1, 1, 1, 1, -1, -1}, {1, 1,
      1, 1, -1, 1}, {1, 1, 1, 1, 1, -1}, {1, 1, 1, 1, 1, 1}};
  hipMemcpyToSymbol(HIP_SYMBOL(cHel), tHel, ncomb * nexternal * sizeof(int)); 
  // perm - nodim
  // static int perm[nexternal] = {0, 1, 2, 3};
}

CPPProcess::~CPPProcess() {}

const std::vector<double> &CPPProcess::getMasses() const {return mME;}

//--------------------------------------------------------------------------
// Initialize process.

void CPPProcess::initProc(string param_card_name) 
{
  // Instantiate the model class and set parameters that stay fixed during run
  pars = Parameters_sm::getInstance(); 
  SLHAReader slha(param_card_name); 
  pars->setIndependentParameters(slha); 
  pars->setIndependentCouplings(); 
  pars->printIndependentParameters(); 
  pars->printIndependentCouplings(); 
  pars->setDependentParameters(); 
  pars->setDependentCouplings(); 
  // Set external particle masses for this matrix element
  mME.push_back(pars->ZERO); 
  mME.push_back(pars->ZERO); 
  mME.push_back(pars->mdl_MT); 
  mME.push_back(pars->mdl_MT); 
  mME.push_back(pars->ZERO); 
  mME.push_back(pars->ZERO); 
  static thrust::complex<double> tIPC[3] = {pars->GC_10, pars->GC_11,
      pars->GC_12};
  static double tIPD[2] = {pars->mdl_MT, pars->mdl_WT}; 

  hipMemcpyToSymbol(HIP_SYMBOL(cIPC), tIPC, 3 * sizeof(thrust::complex<double> )); 
  hipMemcpyToSymbol(HIP_SYMBOL(cIPD), tIPD, 2 * sizeof(double)); 
}

//--------------------------------------------------------------------------
// Evaluate |M|^2, part independent of incoming flavour.

__global__ void sigmaKin(double * allmomenta, double * output) 
{
  // Set the parameters which change event by event
  // Need to discuss this with Stefan
  // pars->setDependentParameters();
  // pars->setDependentCouplings();

  // Reset color flows

  // for (int xx = 0; xx < 384; ++xx) {
  const int nprocesses = 1; 
  int tid = blockIdx.x * blockDim.x + threadIdx.x; 

  // char *devPtr = (char *)tp.ptr;
  // size_t dpt = tp.pitch;
  // size_t slicePitch = dpt * 6;

  // char *dps = devPtr + dim * slicePitch;
  double matrix_element[nprocesses]; 

  thrust::complex<double> amp[159]; 

  double local_m[6][3]; 
  int DIM = blockDim.x * gridDim.x; 
  // for (int i=0; i<20;i++){
  // printf(" %f ", allmomenta[i]);
  // }
  // printf("\n");
  // printf("DIM is %i/%i\n", tid, DIM);
  for (int i = 0; i < 6; i++ )
  {
    for (int j = 0; j < 3; j++ )
    {
      local_m[i][j] = allmomenta[i * 3 * DIM + j * DIM + tid]; 
      // printf(" %f ", local_m[i][j]);
    }
    // printf("\n");
  }


  // Local variables and constants
  const int ncomb = 64; 
  // static bool goodhel[ncomb] = {ncomb * false};
  // static int ntry = 0, sum_hel = 0, ngood = 0;
  // static int igood[ncomb];
  // static int jhel;
  // std::complex<double> **wfs;
  // double t[1];
  // Helicities for the process
  // static const int helicities[ncomb][nexternal] =
  // {{-1,-1,-1,-1,-1,-1},{-1,-1,-1,-1,-1,1},{-1,-1,-1,-1,1,-1},{-1,-1,-1,-1,1,1
  // },{-1,-1,-1,1,-1,-1},{-1,-1,-1,1,-1,1},{-1,-1,-1,1,1,-1},{-1,-1,-1,1,1,1},{
  // -1,-1,1,-1,-1,-1},{-1,-1,1,-1,-1,1},{-1,-1,1,-1,1,-1},{-1,-1,1,-1,1,1},{-1,
  // -1,1,1,-1,-1},{-1,-1,1,1,-1,1},{-1,-1,1,1,1,-1},{-1,-1,1,1,1,1},{-1,1,-1,-1
  // ,-1,-1},{-1,1,-1,-1,-1,1},{-1,1,-1,-1,1,-1},{-1,1,-1,-1,1,1},{-1,1,-1,1,-1,
  // -1},{-1,1,-1,1,-1,1},{-1,1,-1,1,1,-1},{-1,1,-1,1,1,1},{-1,1,1,-1,-1,-1},{-1
  // ,1,1,-1,-1,1},{-1,1,1,-1,1,-1},{-1,1,1,-1,1,1},{-1,1,1,1,-1,-1},{-1,1,1,1,-
  // 1,1},{-1,1,1,1,1,-1},{-1,1,1,1,1,1},{1,-1,-1,-1,-1,-1},{1,-1,-1,-1,-1,1},{1
  // ,-1,-1,-1,1,-1},{1,-1,-1,-1,1,1},{1,-1,-1,1,-1,-1},{1,-1,-1,1,-1,1},{1,-1,-
  // 1,1,1,-1},{1,-1,-1,1,1,1},{1,-1,1,-1,-1,-1},{1,-1,1,-1,-1,1},{1,-1,1,-1,1,-
  // 1},{1,-1,1,-1,1,1},{1,-1,1,1,-1,-1},{1,-1,1,1,-1,1},{1,-1,1,1,1,-1},{1,-1,1
  // ,1,1,1},{1,1,-1,-1,-1,-1},{1,1,-1,-1,-1,1},{1,1,-1,-1,1,-1},{1,1,-1,-1,1,1}
  // ,{1,1,-1,1,-1,-1},{1,1,-1,1,-1,1},{1,1,-1,1,1,-1},{1,1,-1,1,1,1},{1,1,1,-1,
  // -1,-1},{1,1,1,-1,-1,1},{1,1,1,-1,1,-1},{1,1,1,-1,1,1},{1,1,1,1,-1,-1},{1,1,
  // 1,1,-1,1},{1,1,1,1,1,-1},{1,1,1,1,1,1}};
  // Denominators: spins, colors and identical particles
  const int denominators[1] = {512}; 


  // Reset the matrix elements
  for(int i = 0; i < nprocesses; i++ )
  {
    matrix_element[i] = 0.; 
  }
  // Define permutation
  // int perm[nexternal];
  // for(int i = 0; i < nexternal; i++){
  // perm[i]=i;
  // }


  for (int ihel = 0; ihel < ncomb; ihel++ )
  {
    calculate_wavefunctions(ihel, local_m, matrix_element[0]); 
  }


  for (int i = 0; i < nprocesses; ++ i)
  {
    matrix_element[i] /= denominators[i]; 
  }
  for (int i = 0; i < nprocesses; ++ i)
  {
    output[i * nprocesses + tid] = matrix_element[i]; 
    // printf("output %i %i %i %f", tid, i, i*nprocesses+tid,
    // output[i*nprocesses+tid]);

  }


}

//==========================================================================
// Private class member functions

//--------------------------------------------------------------------------


